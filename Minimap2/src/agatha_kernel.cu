#include "hip/hip_runtime.h"
#include "agatha_kernel.h"
#include "kernel_wrapper.h"

#define AGATHA_WARP_NUM 4
#define AGATHA_THREAD_NUM 256
#define SHARED_SIZE AGATHA_THREAD_NUM / AGATHA_WARP_NUM

__global__ void agatha_sort(uint32_t *packed_query_batch, uint32_t *packed_ref_batch,  uint32_t *query_batch_lens, uint32_t *target_batch_lens, uint32_t *query_batch_offsets, uint32_t *target_batch_offsets, int n_tasks, uint32_t max_query_len, short2 *global_buffer_top)
{

    const uint32_t tid = (blockIdx.x * blockDim.x) + threadIdx.x;//thread ID

	uint32_t query_len, ref_len, packed_query_len, packed_ref_len;

	short2* global_ub_idx = (short2*)(global_buffer_top+max_query_len*(blockDim.x/8)*gridDim.x*3);

	if (tid < n_tasks) {

		query_len = query_batch_lens[tid];
		ref_len = target_batch_lens[tid];
		packed_query_len = (query_len >> 3) + (query_len & 7 ? 1 : 0);//number of 32-bit words holding query_batch sequence
		packed_ref_len = (ref_len >> 3) + (ref_len & 7 ? 1 : 0);

		global_ub_idx[tid] = make_short2((packed_ref_len + packed_query_len-1), static_cast<int16_t>(tid));
	}
	
	return;


}


__global__ void agatha_kernel(uint32_t *packed_query_batch, uint32_t *packed_ref_batch,  uint32_t *query_batch_lens, uint32_t *target_batch_lens, uint32_t *query_batch_offsets, uint32_t *target_batch_offsets, gasal_res_t *device_res, gasal_res_t *device_res_second, uint4 *packed_tb_matrices, int n_tasks, uint32_t max_query_len, short2 *global_buffer_top)
{
    /*Initial kernel setup*/

	int bw = BW_EXTEND;
	// Initializing variables 
	int32_t i, k, m, l, y, e;
	int32_t ub_idx, job_idx, ref_idx, query_idx;
	short2 HD;
	int32_t temp_score;
	int slice_start, slice_end, finished_blocks, chunk_start, chunk_end;
	int packed_ref_idx, packed_query_idx;
	int total_anti_diags;
	register uint32_t packed_ref_literal, packed_query_literal; 
	bool active, terminated;
	int32_t packed_ref_batch_idx, packed_query_batch_idx, query_len, ref_len, packed_query_len, packed_ref_len;
	int diag_idx, temp, last_diag;
	int test_id;

	// Initializing max score and its idx
    int32_t max_score = 0; 
	int32_t max_ref_idx = 0; 
    int32_t prev_max_score = 0;
    int32_t max_query_idx = 0;

	// Setting constant values
	const short2 initHD = make_short2(MINUS_INF2, MINUS_INF2); //used to initialize short2
	const int32_t tid = (blockIdx.x * blockDim.x) + threadIdx.x; //thread ID within the entire kernel
	const int packed_len = 8; //number of bps (literals) packed into a single int32
	const int const_warp_len = 8; //number of threads per subwarp (before subwarp rejoining occurs)
	const int real_warp_id = threadIdx.x % 32; //thread ID within a single (full 32-thread) warp
	const int warp_per_kernel = (gridDim.x * blockDim.x) / const_warp_len; // number of subwarps. assume number of threads % const_warp_len == 0
	const int job_per_warp = n_tasks % warp_per_kernel ? (n_tasks / warp_per_kernel + 1) : n_tasks / warp_per_kernel; //number of jobs (alignments/tasks) needed to be done by a single subwarp
	const int job_per_query = max_query_len % const_warp_len ? (max_query_len / const_warp_len + 1) : max_query_len / const_warp_len; //number of a literal's initial score to fill per thread
	const int job_start_idx = (tid / const_warp_len)*job_per_warp; // the boundary of jobs of a subwarp 
	const int job_end_idx = (job_start_idx + job_per_warp) < n_tasks ? (job_start_idx + job_per_warp) : n_tasks; // the boundary of jobs of a subwarp
	const int total_shm = packed_len*(_cudaSliceWidth+1); // amount of shared memory a single thread uses
	
	// Arrays for saving intermediate values
	int32_t h[9];
	int32_t f[9];
	int32_t p[9];

	// Global memory setup
	short2* global_buffer_left = (short2*)(global_buffer_top+max_query_len*(blockDim.x/8)*gridDim.x);
	int32_t* global_buffer_topleft= (int32_t*)(global_buffer_left+max_query_len*(blockDim.x/8)*gridDim.x);
	short2* global_ub_idx = (short2*)(global_buffer_top+max_query_len*(blockDim.x/8)*gridDim.x*3);

	// Shared memory setup
	extern __shared__ int32_t shared_maxHH[];
	int32_t* antidiag_max = (int32_t*)(shared_maxHH+(threadIdx.x/32)*total_shm*32);
	int32_t* shared_job = shared_maxHH+(blockDim.x/32)*total_shm*32+(threadIdx.x/32)*28;

	/* Setup values that will change after Subwarp Rejoining */
	int warp_len = const_warp_len;
	int warp_id = threadIdx.x % warp_len; // id of a thread in a subwarp 
	int warp_num = tid / warp_len;
	// mask that is true for threads in the same subwarp
	unsigned same_threads = __match_any_sync(0xffffffff, warp_num);
	if (warp_id==0) shared_job[(warp_num&3)] = -1;

	/* Iterating over jobs/alignments */
	for (job_idx = job_start_idx; job_idx < job_end_idx; job_idx++) {
		
		/*Uneven Bucketing*/
		// the first subwarp fetches a long sequence's idx, while the remaining subwarps fetch short sequences' idx
		ub_idx = ((job_idx&3)==0)? global_ub_idx[n_tasks-(job_idx>>2)-1].y: global_ub_idx[job_idx-(job_idx>>2)-1].y;
		
		// get target and query sequence information
		packed_ref_batch_idx = target_batch_offsets[ub_idx] >> 3; //starting index of the target_batch sequence
		packed_query_batch_idx = query_batch_offsets[ub_idx] >> 3;//starting index of the query_batch sequence
		query_len = query_batch_lens[ub_idx]; // query sequence length
		ref_len = target_batch_lens[ub_idx]; // reference sequence length 
		packed_query_len = (query_len >> 3) + (query_len & 7 ? 1 : 0);//number of 32-bit words holding query_batch sequence
		packed_ref_len = (ref_len >> 3) + (ref_len & 7 ? 1 : 0);//number of 32-bit words holding target_batch sequence

		// check if alignment in the chain is done 


		//if(warp_id==0) printf("u_id: %d | ofs: r(%d), q(%d), len: r(%d), q(%d)\n", ub_idx, packed_ref_batch_idx, packed_query_batch_idx, query_len, ref_len);
		/*Buffer Initialization*/
		// fill global buffer with initial value
		// global_buffer_top: used to store intermediate scores H and E in the horizontal strip (scores from the top)
		for (i = 0; i < job_per_query; i++) {
			l = i*warp_len + warp_id;
			if ((l) < max_query_len) {
				k = -(_cudaGapOE + (_cudaGapExtend*(l)));
				global_buffer_top[warp_num*max_query_len + l] =  l <= bw? make_short2(k, k-_cudaGapOE):initHD;	
			}
		}
		// global_buffer_left: used to store intermediate scores H and F in the vertical strip (scores from the left)
		for (i = 0; i < job_per_query; i++) {
			l = i*warp_len + warp_id;
			if ((l) < max_query_len) {
				k = -(_cudaGapOE + (_cudaGapExtend*(l)));
				global_buffer_left[warp_num*max_query_len + l] =  l <= bw? make_short2(k, k-_cudaGapOE):initHD;	
			}
		}
		// global_buffer_topleft: used to store intermediate scores H in the diagonal strip (scores from the top-left)
		for (i = 0; i < job_per_query; i++) {
			l = i*warp_len + warp_id;
			if (l < max_query_len) {
				k = -(_cudaGapOE+(_cudaGapExtend*(l*packed_len-1)));
				global_buffer_topleft[warp_num*max_query_len + l] = l==0? 0: (l*packed_len-1) <= bw? k: MINUS_INF2; 	
			}
		}
		// fill shared memory with initial value
		for (m = 0; m < total_shm; m++) {
			antidiag_max[real_warp_id + m*32] = INT_MIN;
		}

		__syncwarp();

		// Initialize variables
		max_score = 0; 
		prev_max_score = 0;
		max_ref_idx = 0; 
    	max_query_idx = 0;
		terminated = false;

		// check termination condition

		i = 0; //chunk
		total_anti_diags = packed_ref_len + packed_query_len-1; //chunk

		/*Subwarp Rejoining*/
		//set shared memory that is used to maintain values for subwarp rejoining
		if (warp_id==0) shared_job[(warp_num&3)] = total_anti_diags;
		else if (warp_id==1) shared_job[4+(warp_num&3)] = packed_ref_batch_idx;
		else if (warp_id==2) shared_job[8+(warp_num&3)] = packed_query_batch_idx;
		else if (warp_id==3) shared_job[12+(warp_num&3)] = (ref_len<<16)+query_len;
		else if (warp_id==4) shared_job[16+(warp_num&3)] = ub_idx;

		same_threads = __match_any_sync(__activemask(), warp_num);

		__syncwarp();

		/*Main Alignment Loop*/
		while (i < total_anti_diags) {
			
			// set boundaries for current slice
			slice_start = max(0, (i-packed_query_len+1));
			slice_start = max(slice_start, (i*packed_len + packed_len-1+1 - bw)/2/packed_len);
			slice_end = min(packed_ref_len-1, i+_cudaSliceWidth-1);
			slice_end = min(slice_end, ((i+_cudaSliceWidth-1)*packed_len + packed_len-1 + bw)/2/packed_len);
			finished_blocks = slice_start;
			
			if (slice_start > slice_end) {
				terminated = true;
			}

			while (!terminated && finished_blocks <= slice_end) {
				// while the entire chunk diag is not finished
				packed_ref_idx = finished_blocks + warp_id;
				packed_query_idx = i - packed_ref_idx;
				active = (packed_ref_idx <= slice_end);	//whether the current thread has cells to fill or not
				
				if (active) {
					ref_idx = packed_ref_idx << 3;
					query_idx = packed_query_idx << 3;

					// load intermediate values from global buffers
					p[1] = global_buffer_topleft[warp_num*max_query_len + packed_ref_idx];

					for (m = 1; m < 9; m++) {
						if ( (ref_idx + m-1) < ref_len) {
							HD = global_buffer_left[warp_num*max_query_len + ref_idx + m-1];
							h[m] = HD.x;
							f[m] = HD.y;
						} else {
							// if index out of bound of the score table 
							h[m] = MINUS_INF2;
							f[m] = MINUS_INF2;
						}
						
					}

					for (m=2;m<9;m++) {
						p[m] = h[m-1];
					}

					// Set boundaries for the current chunk
					chunk_start = (max(0, (packed_ref_idx*packed_len - bw)))/packed_len;
					chunk_end = min( packed_query_len-1, ( (packed_ref_idx*packed_len + packed_len -1 + bw)) /packed_len );
					packed_ref_literal = packed_ref_batch[packed_ref_batch_idx + packed_ref_idx];
				}
					
				// Compute the current chunk
				for (y = 0; y < _cudaSliceWidth; y++) {
					if (active && chunk_start <= packed_query_idx && packed_query_idx <= chunk_end) {
						packed_query_literal = packed_query_batch[packed_query_batch_idx + packed_query_idx]; 
						query_idx = packed_query_idx << 3;
						
						for (k = 28; k >= 0 && query_idx < query_len; k -= 4) {
							uint32_t qbase = (packed_query_literal >> k) & 15;	//get a base from query_batch sequence
							// load intermediate values from global buffers
							HD = global_buffer_top[warp_num*max_query_len + query_idx];
							h[0] = HD.x;
							e = HD.y;

							if (packed_query_idx == chunk_start || packed_query_idx == chunk_end) {
								#pragma unroll 8
								for (l = 28, m = 1; m < 9; l -= 4, m++) {
									CORE_COMPUTE_BOUNDARY();
								}
							} else {
								#pragma unroll 8
								for (l = 28, m = 1; m < 9; l -= 4, m++) {
									CORE_COMPUTE();
								}
							}
							
							// write intermediate values to global buffers
							HD.x = h[m-1];
							HD.y = e;
							global_buffer_top[warp_num*max_query_len + query_idx] = HD;

							query_idx++;

						}

					}
					

					packed_query_idx++;
					
				}
				
				// write intermediate values to global buffers
				if (active) {	
					for (m = 1; m < 9; m++) {
						if ( ref_idx + m-1 < ref_len) {
							HD.x = h[m];
							HD.y = f[m];
							global_buffer_left[warp_num*max_query_len + ref_idx + m-1] = HD;
						}
					}
					global_buffer_topleft[warp_num*max_query_len + packed_ref_idx] = p[1];
				}
				
				finished_blocks+=warp_len;
			}

			__syncwarp();

			last_diag = (i+_cudaSliceWidth)<<3;
			prev_max_score = query_len+ref_len-1;

			/* Termination Condition & Score Update */
			if (!terminated) {
				for (diag_idx = i<<3; diag_idx < last_diag; diag_idx++) {
					if (diag_idx <prev_max_score) {
						m = diag_idx&(total_shm-1);
						temp = __reduce_max_sync(same_threads, antidiag_max[(m<<5)+real_warp_id]);
						if ((temp>>16) > max_score) {				
							max_score = temp>>16;
							max_ref_idx = (temp&65535);
							max_query_idx = diag_idx-max_ref_idx; 
						} else if ( (temp&65535) >= max_ref_idx && (diag_idx-(temp&65535)) >= max_query_idx) {
							int tl =  (temp&65535) - max_ref_idx, ql = (diag_idx-(temp&65535)) - max_query_idx, l;
							l = tl > ql? tl - ql : ql - tl;
							if (_cudaZThreshold >= 0 && max_score - (temp>>16) > _cudaZThreshold + l*_cudaGapExtend) {
								// Termination condition is met
								// write zdrop information to global drop array
								terminated = true;
								break;
							}
						}
						// reset shared memory buffer for next slice
						antidiag_max[(m<<5)+real_warp_id]=INT_MIN;
					}
				}
			}
			
			__syncwarp();

			// If job is finished
			if (terminated) {
				total_anti_diags = i; // set the total amount of diagonals as the current diagonal (to indicate that the job has finished)	
				if (warp_id==0) shared_job[(warp_num&3)] = total_anti_diags; //update this to shared memory as well (this will be used in Subwarp Rejoining as an indicator that the subwarp's job is done)
			}
			
			// Update the max score and its index to shared memory (used in Subwarp Rejoining)
			if (warp_id==1) shared_job[20+(warp_num&3)] = max_score;
			else if (warp_id==2) shared_job[24+(warp_num&3)] = (max_ref_idx<<16) + max_query_idx;
 
			__syncwarp();

			i += _cudaSliceWidth;

			/*Job wrap-up*/
			// If the job is done (either due to (1) meeting the termination condition (2) all the diagonals have been computed)
			if (i >= total_anti_diags) {
				
				// In the case of (2), check the termination condition & score update for the last diagonal block
				if (!terminated) {
					diag_idx = (i*packed_len)&(total_shm-1);
					for (k = i*packed_len, m = diag_idx; m < diag_idx+packed_len; m++, k++) {
						temp = __reduce_max_sync(same_threads, antidiag_max[(m<<5)+real_warp_id]);
						if ((temp>>16) > max_score) {				
							max_score = temp>>16;
							max_ref_idx = (temp&65535);
							max_query_idx = k-max_ref_idx; 
						} else if ( (temp&65535) >= max_ref_idx && (k-(temp&65535)) >= max_query_idx) {
							int tl =  (temp&65535) - max_ref_idx, ql = (k-(temp&65535)) - max_query_idx, l;
							l = tl > ql? tl - ql : ql - tl;
							if (_cudaZThreshold >= 0 && max_score - (temp>>16) > _cudaZThreshold + l*_cudaGapExtend) {
								// Termination condition is met
								terminated = true;
								break;
							}
						}
						antidiag_max[(m<<5)+real_warp_id]=INT_MIN;
					}
				}
				
				// Spill the results to GPU memory to be later moved to the CPU
				if (warp_id==0) {
					device_res->aln_score[ub_idx] = max_score;//copy the max score to the output array in the GPU mem
					device_res->query_batch_end[ub_idx] = max_query_idx;//copy the end position on query_batch sequence to the output array in the GPU mem
					device_res->target_batch_end[ub_idx] = max_ref_idx;//copy the end position on target_batch sequence to the output array in the GPU mem
					//if(max_score > 0)
					// printf("[idx: %d] score: %d, q_end: %d, t_end: %d, qlen: %d, rlen: %d\n", ub_idx,max_score, max_query_idx, max_ref_idx, query_len, ref_len);
				}

				/*Subwarp Rejoining*/
				// The subwarp that has no job looks for new jobs by iterating over other subwarp's job
				for (m = 0; m < (32/const_warp_len); m++) {
					// if the selected job still has remainig diagonals
					if (shared_job[m] > i) { // possible because all subwarps sync after each diagonal block is finished
						// read the selected job's info
						total_anti_diags = shared_job[m];
						warp_num = ((warp_num>>2)<<2)+m;
						ub_idx = shared_job[16+m];

						packed_ref_batch_idx = shared_job[4+m];
						packed_query_batch_idx = shared_job[8+m];
						ref_len = shared_job[12+m];
						query_len = ref_len&65535;
						ref_len = ref_len>>16;
						packed_query_len = (query_len >> 3) + (query_len & 7 ? 1 : 0);
						packed_ref_len = (ref_len >> 3) + (ref_len & 7 ? 1 : 0);
						
						max_score = shared_job[20+m];
						max_ref_idx = shared_job[24+m];
						max_query_idx = max_ref_idx&65535;
						max_ref_idx = max_ref_idx>>16;
						
						// reset the flag
						terminated = false;

						// reset shared memory buffer
						for (m = 0; m < total_shm; m++) {
							antidiag_max[(m<<5)+real_warp_id]=INT_MIN;
						}
						
						break;
					}
				}

			}

			__syncwarp();
			
			/*Subwarp Rejoining*/
			//Set the mask, warp length and thread id within the warp 
			same_threads = __match_any_sync(__activemask(), warp_num);
			warp_len = __popc(same_threads);
			warp_id = __popc((((0xffffffff) << (threadIdx.x % 32))&same_threads))-1;
			
			__syncwarp();

		}
		__syncwarp();
		/*Subwarp Rejoining*/
		//Reset subwarp and job related values for the next iteration
		warp_len = const_warp_len;
		warp_num = tid / warp_len;
		warp_id = tid % const_warp_len;
		ub_idx = shared_job[16+(warp_num&3)];

		__syncwarp();



	}
	
	return;


}

__global__ void agatha_kernel_static_tb(uint32_t *packed_query_batch, uint32_t *packed_ref_batch,  uint32_t *query_batch_lens, uint32_t *target_batch_lens, uint32_t *query_batch_offsets, uint32_t *target_batch_offsets, gasal_res_t *device_res, gasal_res_t *device_res_second, uint32_t *packed_tb_matrices, int n_tasks, uint32_t max_query_len, short2 *global_buffer_top)
{
    /*Initial kernel setup*/

	int bw = BW_EXTEND;
	// Initializing variables 
	int32_t i, k, m, l, y, e;
	int32_t ub_idx, job_idx, ref_idx, query_idx;
	short2 HD;
	int32_t temp_score;
	int slice_start, slice_end, finished_blocks, chunk_start, chunk_end;
	int packed_ref_idx, packed_query_idx;
	int total_anti_diags;
	register uint32_t packed_ref_literal, packed_query_literal; 
	bool active, terminated;
	int32_t packed_ref_batch_idx, packed_query_batch_idx, query_len, ref_len, packed_query_len, packed_ref_len;
	int diag_idx, temp, last_diag;
	int test_id;

	// Initializing max score and its idx
    int32_t max_score = 0; 
	int32_t max_ref_idx = 0; 
    int32_t prev_max_score = 0;
    int32_t max_query_idx = 0;

	// Setting constant values
	const short2 initHD = make_short2(MINUS_INF2, MINUS_INF2); //used to initialize short2
	const int32_t tid = (blockIdx.x * blockDim.x) + threadIdx.x; //thread ID within the entire kernel
	const int packed_len = 8; //number of bps (literals) packed into a single int32
	const int const_warp_len = 8; //number of threads per subwarp (before subwarp rejoining occurs)
	const int real_warp_id = threadIdx.x % 32; //thread ID within a single (full 32-thread) warp
	const int warp_per_kernel = (gridDim.x * blockDim.x) / const_warp_len; // number of subwarps. assume number of threads % const_warp_len == 0
	const int job_per_warp = n_tasks % warp_per_kernel ? (n_tasks / warp_per_kernel + 1) : n_tasks / warp_per_kernel; //number of jobs (alignments/tasks) needed to be done by a single subwarp
	const int job_per_query = max_query_len % const_warp_len ? (max_query_len / const_warp_len + 1) : max_query_len / const_warp_len; //number of a literal's initial score to fill per thread
	const int job_start_idx = (tid / const_warp_len)*job_per_warp; // the boundary of jobs of a subwarp 
	const int job_end_idx = (job_start_idx + job_per_warp) < n_tasks ? (job_start_idx + job_per_warp) : n_tasks; // the boundary of jobs of a subwarp
	const int total_shm = packed_len*(_cudaSliceWidth+1); // amount of shared memory a single thread uses
	
	// Arrays for saving intermediate values
	int32_t h[9];
	int32_t f[9];
	int32_t p[9];

	// Global memory setup
	short2* global_buffer_left = (short2*)(global_buffer_top+max_query_len*(blockDim.x/8)*gridDim.x);
	int32_t* global_buffer_topleft= (int32_t*)(global_buffer_left+max_query_len*(blockDim.x/8)*gridDim.x);
	short2* global_ub_idx = (short2*)(global_buffer_top+max_query_len*(blockDim.x/8)*gridDim.x*3);

	// Shared memory setup
	extern __shared__ int32_t shared_maxHH[];
	int32_t* antidiag_max = (int32_t*)(shared_maxHH+(threadIdx.x/32)*total_shm*32);
	int32_t* shared_job = shared_maxHH+(blockDim.x/32)*total_shm*32+(threadIdx.x/32)*28;

	/* Setup values that will change after Subwarp Rejoining */
	int warp_len = const_warp_len;
	int warp_id = threadIdx.x % warp_len; // id of a thread in a subwarp 
	int warp_num = tid / warp_len;
	// mask that is true for threads in the same subwarp
	unsigned same_threads = __match_any_sync(0xffffffff, warp_num);
	if (warp_id==0) shared_job[(warp_num&3)] = -1;

	/* Iterating over jobs/alignments */
	for (job_idx = job_start_idx; job_idx < job_end_idx; job_idx++) {
		
		/*Uneven Bucketing*/
		// the first subwarp fetches a long sequence's idx, while the remaining subwarps fetch short sequences' idx
		ub_idx = ((job_idx&3)==0)? global_ub_idx[n_tasks-(job_idx>>2)-1].y: global_ub_idx[job_idx-(job_idx>>2)-1].y;
		
		// get target and query sequence information
		packed_ref_batch_idx = target_batch_offsets[ub_idx] >> 3; //starting index of the target_batch sequence
		packed_query_batch_idx = query_batch_offsets[ub_idx] >> 3;//starting index of the query_batch sequence
		query_len = query_batch_lens[ub_idx]; // query sequence length
		ref_len = target_batch_lens[ub_idx]; // reference sequence length 
		packed_query_len = (query_len >> 3) + (query_len & 7 ? 1 : 0);//number of 32-bit words holding query_batch sequence
		packed_ref_len = (ref_len >> 3) + (ref_len & 7 ? 1 : 0);//number of 32-bit words holding target_batch sequence

		// check if alignment in the chain is done 
	
		/*Buffer Initialization*/
		// fill global buffer with initial value
		// global_buffer_top: used to store intermediate scores H and E in the horizontal strip (scores from the top)
		for (i = 0; i < job_per_query; i++) {
			l = i*warp_len + warp_id;
			if ((l) < max_query_len) {
				k = -(_cudaGapOE + (_cudaGapExtend*(l)));
				global_buffer_top[warp_num*max_query_len + l] =  l <= bw? make_short2(k, k-_cudaGapOE):initHD;	
			}
		}
		// global_buffer_left: used to store intermediate scores H and F in the vertical strip (scores from the left)
		for (i = 0; i < job_per_query; i++) {
			l = i*warp_len + warp_id;
			if ((l) < max_query_len) {
				k = -(_cudaGapOE + (_cudaGapExtend*(l)));
				global_buffer_left[warp_num*max_query_len + l] =  l <= bw? make_short2(k, k-_cudaGapOE):initHD;	
			}
		}
		// global_buffer_topleft: used to store intermediate scores H in the diagonal strip (scores from the top-left)
		for (i = 0; i < job_per_query; i++) {
			l = i*warp_len + warp_id;
			if (l < max_query_len) {
				k = -(_cudaGapOE+(_cudaGapExtend*(l*packed_len-1)));
				global_buffer_topleft[warp_num*max_query_len + l] = l==0? 0: (l*packed_len-1) <= bw? k: MINUS_INF2; 	
			}
		}
		// fill shared memory with initial value
		for (m = 0; m < total_shm; m++) {
			antidiag_max[real_warp_id + m*32] = INT_MIN;
		}

		__syncwarp();

		// Initialize variables
		max_score = 0; 
		prev_max_score = 0;
		max_ref_idx = 0; 
    	max_query_idx = 0;
		terminated = false;

		// check termination condition

		i = 0; //chunk
		total_anti_diags = packed_ref_len + packed_query_len-1; //chunk

		/*Subwarp Rejoining*/
		//set shared memory that is used to maintain values for subwarp rejoining
		if (warp_id==0) shared_job[(warp_num&3)] = total_anti_diags;
		else if (warp_id==1) shared_job[4+(warp_num&3)] = packed_ref_batch_idx;
		else if (warp_id==2) shared_job[8+(warp_num&3)] = packed_query_batch_idx;
		else if (warp_id==3) shared_job[12+(warp_num&3)] = (ref_len<<16)+query_len;
		else if (warp_id==4) shared_job[16+(warp_num&3)] = ub_idx;

		same_threads = __match_any_sync(__activemask(), warp_num);

		__syncwarp();

		/*Main Alignment Loop*/
		while (i < total_anti_diags) {
			
			// set boundaries for current slice
			slice_start = max(0, (i-packed_query_len+1));
			slice_start = max(slice_start, (i*packed_len + packed_len-1+1 - bw)/2/packed_len);
			slice_end = min(packed_ref_len-1, i+_cudaSliceWidth-1);
			slice_end = min(slice_end, ((i+_cudaSliceWidth-1)*packed_len + packed_len-1 + bw)/2/packed_len);
			finished_blocks = slice_start;
			
			if (slice_start > slice_end) {
				terminated = true;
			}

			while (!terminated && finished_blocks <= slice_end) {
				// while the entire chunk diag is not finished
				packed_ref_idx = finished_blocks + warp_id;
				packed_query_idx = i - packed_ref_idx;
				active = (packed_ref_idx <= slice_end);	//whether the current thread has cells to fill or not
				
				if (active) {
					ref_idx = packed_ref_idx << 3;
					query_idx = packed_query_idx << 3;

					// load intermediate values from global buffers
					p[1] = global_buffer_topleft[warp_num*max_query_len + packed_ref_idx];

					for (m = 1; m < 9; m++) {
						if ( (ref_idx + m-1) < ref_len) {
							HD = global_buffer_left[warp_num*max_query_len + ref_idx + m-1];
							h[m] = HD.x;
							f[m] = HD.y;
						} else {
							// if index out of bound of the score table 
							h[m] = MINUS_INF2;
							f[m] = MINUS_INF2;
						}
						
					}

					for (m=2;m<9;m++) {
						p[m] = h[m-1];
					}

					// Set boundaries for the current chunk
					chunk_start = (max(0, (packed_ref_idx*packed_len - bw)))/packed_len;
					chunk_end = min( packed_query_len-1, ( (packed_ref_idx*packed_len + packed_len -1 + bw)) /packed_len );
					packed_ref_literal = packed_ref_batch[packed_ref_batch_idx + packed_ref_idx];
				}
					
				// Compute the current chunk
				for (y = 0; y < _cudaSliceWidth; y++) {
					if (active && chunk_start <= packed_query_idx && packed_query_idx <= chunk_end) {
						packed_query_literal = packed_query_batch[packed_query_batch_idx + packed_query_idx]; 
						query_idx = packed_query_idx << 3;
						
						for (k = 28; k >= 0 && query_idx < query_len; k -= 4) {
							uint32_t qbase = (packed_query_literal >> k) & 15;	//get a base from query_batch sequence
							// load intermediate values from global buffers
							HD = global_buffer_top[warp_num*max_query_len + query_idx];
							h[0] = HD.x;
							e = HD.y;
							
							if (packed_query_idx == chunk_start || packed_query_idx == chunk_end) {
								#pragma unroll 8
								for (l = 28, m = 1; m < 9; l -= 4, m++) {
									CORE_COMPUTE_BOUNDARY_TB(packed_tb_matrices[ub_idx*max_query_len*max_query_len/8 + packed_ref_idx * max_query_len + query_idx]);
								}
							} else {
								#pragma unroll 8
								for (l = 28, m = 1; m < 9; l -= 4, m++) {
									CORE_COMPUTE_TB(packed_tb_matrices[ub_idx*max_query_len*max_query_len/8 + packed_ref_idx * max_query_len + query_idx]);
								}
							}
							
							// write intermediate values to global buffers
							HD.x = h[m-1];
							HD.y = e;
							global_buffer_top[warp_num*max_query_len + query_idx] = HD;

							query_idx++;

						}

					}
					

					packed_query_idx++;
					
				}
				
				// write intermediate values to global buffers
				if (active) {	
					for (m = 1; m < 9; m++) {
						if ( ref_idx + m-1 < ref_len) {
							HD.x = h[m];
							HD.y = f[m];
							global_buffer_left[warp_num*max_query_len + ref_idx + m-1] = HD;
						}
					}
					global_buffer_topleft[warp_num*max_query_len + packed_ref_idx] = p[1];
				}
				
				finished_blocks+=warp_len;
			}

			__syncwarp();

			last_diag = (i+_cudaSliceWidth)<<3;
			prev_max_score = query_len+ref_len-1;

			/* Termination Condition & Score Update */
			if (!terminated) {
				for (diag_idx = i<<3; diag_idx < last_diag; diag_idx++) {
					if (diag_idx <prev_max_score) {
						m = diag_idx&(total_shm-1);
						temp = __reduce_max_sync(same_threads, antidiag_max[(m<<5)+real_warp_id]);
						if ((temp>>16) > max_score) {				
							max_score = temp>>16;
							max_ref_idx = (temp&65535);
							max_query_idx = diag_idx-max_ref_idx; 
						} else if ( (temp&65535) >= max_ref_idx && (diag_idx-(temp&65535)) >= max_query_idx) {
							int tl =  (temp&65535) - max_ref_idx, ql = (diag_idx-(temp&65535)) - max_query_idx, l;
							l = tl > ql? tl - ql : ql - tl;
							if (_cudaZThreshold >= 0 && max_score - (temp>>16) > _cudaZThreshold + l*_cudaGapExtend) {
								// Termination condition is met
								// write zdrop information to global drop array
								terminated = true;
								break;
							}
						}
						// reset shared memory buffer for next slice
						antidiag_max[(m<<5)+real_warp_id]=INT_MIN;
					}
				}
			}
			
			__syncwarp();

			// If job is finished
			if (terminated) {
				total_anti_diags = i; // set the total amount of diagonals as the current diagonal (to indicate that the job has finished)	
				if (warp_id==0) shared_job[(warp_num&3)] = total_anti_diags; //update this to shared memory as well (this will be used in Subwarp Rejoining as an indicator that the subwarp's job is done)
			}
			
			// Update the max score and its index to shared memory (used in Subwarp Rejoining)
			if (warp_id==1) shared_job[20+(warp_num&3)] = max_score;
			else if (warp_id==2) shared_job[24+(warp_num&3)] = (max_ref_idx<<16) + max_query_idx;
 
			__syncwarp();

			i += _cudaSliceWidth;

			/*Job wrap-up*/
			// If the job is done (either due to (1) meeting the termination condition (2) all the diagonals have been computed)
			if (i >= total_anti_diags) {
				
				// In the case of (2), check the termination condition & score update for the last diagonal block
				if (!terminated) {
					diag_idx = (i*packed_len)&(total_shm-1);
					for (k = i*packed_len, m = diag_idx; m < diag_idx+packed_len; m++, k++) {
						temp = __reduce_max_sync(same_threads, antidiag_max[(m<<5)+real_warp_id]);
						if ((temp>>16) > max_score) {				
							max_score = temp>>16;
							max_ref_idx = (temp&65535);
							max_query_idx = k-max_ref_idx; 
						} else if ( (temp&65535) >= max_ref_idx && (k-(temp&65535)) >= max_query_idx) {
							int tl =  (temp&65535) - max_ref_idx, ql = (k-(temp&65535)) - max_query_idx, l;
							l = tl > ql? tl - ql : ql - tl;
							if (_cudaZThreshold >= 0 && max_score - (temp>>16) > _cudaZThreshold + l*_cudaGapExtend) {
								// Termination condition is met
								terminated = true;
								break;
							}
						}
						antidiag_max[(m<<5)+real_warp_id]=INT_MIN;
					}
				}
				
				// Spill the results to GPU memory to be later moved to the CPU
				if (warp_id==0) {
					device_res->aln_score[ub_idx] = max_score;//copy the max score to the output array in the GPU mem
					device_res->query_batch_end[ub_idx] = max_query_idx;//copy the end position on query_batch sequence to the output array in the GPU mem
					device_res->target_batch_end[ub_idx] = max_ref_idx;//copy the end position on target_batch sequence to the output array in the GPU mem
				}

				/*Subwarp Rejoining*/
				// The subwarp that has no job looks for new jobs by iterating over other subwarp's job
				for (m = 0; m < (32/const_warp_len); m++) {
					// if the selected job still has remainig diagonals
					if (shared_job[m] > i) { // possible because all subwarps sync after each diagonal block is finished
						// read the selected job's info
						total_anti_diags = shared_job[m];
						warp_num = ((warp_num>>2)<<2)+m;
						ub_idx = shared_job[16+m];

						packed_ref_batch_idx = shared_job[4+m];
						packed_query_batch_idx = shared_job[8+m];
						ref_len = shared_job[12+m];
						query_len = ref_len&65535;
						ref_len = ref_len>>16;
						packed_query_len = (query_len >> 3) + (query_len & 7 ? 1 : 0);
						packed_ref_len = (ref_len >> 3) + (ref_len & 7 ? 1 : 0);
						
						max_score = shared_job[20+m];
						max_ref_idx = shared_job[24+m];
						max_query_idx = max_ref_idx&65535;
						max_ref_idx = max_ref_idx>>16;
						
						// reset the flag
						terminated = false;

						// reset shared memory buffer
						for (m = 0; m < total_shm; m++) {
							antidiag_max[(m<<5)+real_warp_id]=INT_MIN;
						}
						
						break;
					}
				}

			}

			__syncwarp();
			
			/*Subwarp Rejoining*/
			//Set the mask, warp length and thread id within the warp 
			same_threads = __match_any_sync(__activemask(), warp_num);
			warp_len = __popc(same_threads);
			warp_id = __popc((((0xffffffff) << (threadIdx.x % 32))&same_threads))-1;
			
			__syncwarp();

		}
		__syncwarp();
		/*Subwarp Rejoining*/
		//Reset subwarp and job related values for the next iteration
		warp_len = const_warp_len;
		warp_num = tid / warp_len;
		warp_id = tid % const_warp_len;
		ub_idx = shared_job[16+(warp_num&3)];

		__syncwarp();



	}
	
	return;


}

__global__ void agatha_kernel_dynamic_tb(uint32_t *packed_query_batch, uint32_t *packed_ref_batch,  uint32_t *query_batch_lens, uint32_t *target_batch_lens, uint32_t *query_batch_offsets, uint32_t *target_batch_offsets, gasal_res_t *device_res, gasal_res_t *device_res_second, short2 *dblock_row, short2 *dblock_col, int n_tasks, uint32_t max_query_len, short2 *global_buffer_top)
{
    /*Initial kernel setup*/

	int bw = BW_EXTEND;
	// Initializing variables 
	int32_t i, k, m, l, y, e;
	int32_t ub_idx, job_idx, ref_idx, query_idx;
	short2 HD;
	int32_t temp_score;
	int slice_start, slice_end, finished_blocks, chunk_start, chunk_end;
	int packed_ref_idx, packed_query_idx;
	int total_anti_diags;
	register uint32_t packed_ref_literal, packed_query_literal; 
	bool active, terminated;
	int32_t packed_ref_batch_idx, packed_query_batch_idx, query_len, ref_len, packed_query_len, packed_ref_len;
	int diag_idx, temp, last_diag;
	int test_id;

	// Initializing max score and its idx
    int32_t max_score = 0; 
	int32_t max_ref_idx = 0; 
    int32_t prev_max_score = 0;
    int32_t max_query_idx = 0;

	// Setting constant values
	const short2 initHD = make_short2(MINUS_INF2, MINUS_INF2); //used to initialize short2
	const int32_t tid = (blockIdx.x * blockDim.x) + threadIdx.x; //thread ID within the entire kernel
	const int packed_len = 8; //number of bps (literals) packed into a single int32
	const int const_warp_len = 8; //number of threads per subwarp (before subwarp rejoining occurs)
	const int real_warp_id = threadIdx.x % 32; //thread ID within a single (full 32-thread) warp
	const int warp_per_kernel = (gridDim.x * blockDim.x) / const_warp_len; // number of subwarps. assume number of threads % const_warp_len == 0
	const int job_per_warp = n_tasks % warp_per_kernel ? (n_tasks / warp_per_kernel + 1) : n_tasks / warp_per_kernel; //number of jobs (alignments/tasks) needed to be done by a single subwarp
	const int job_per_query = max_query_len % const_warp_len ? (max_query_len / const_warp_len + 1) : max_query_len / const_warp_len; //number of a literal's initial score to fill per thread
	const int job_start_idx = (tid / const_warp_len)*job_per_warp; // the boundary of jobs of a subwarp 
	const int job_end_idx = (job_start_idx + job_per_warp) < n_tasks ? (job_start_idx + job_per_warp) : n_tasks; // the boundary of jobs of a subwarp
	const int total_shm = packed_len*(_cudaSliceWidth+1); // amount of shared memory a single thread uses
	
	// Arrays for saving intermediate values
	int32_t h[9];
	int32_t f[9];
	int32_t p[9];

	// Global memory setup
	short2* global_buffer_left = (short2*)(global_buffer_top+max_query_len*(blockDim.x/8)*gridDim.x);
	int32_t* global_buffer_topleft= (int32_t*)(global_buffer_left+max_query_len*(blockDim.x/8)*gridDim.x);
	short2* global_ub_idx = (short2*)(global_buffer_top+max_query_len*(blockDim.x/8)*gridDim.x*3);

	// Shared memory setup
	extern __shared__ int32_t shared_maxHH[];
	int32_t* antidiag_max = (int32_t*)(shared_maxHH+(threadIdx.x/32)*total_shm*32);
	int32_t* shared_job = shared_maxHH+(blockDim.x/32)*total_shm*32+(threadIdx.x/32)*28;

	/* Setup values that will change after Subwarp Rejoining */
	int warp_len = const_warp_len;
	int warp_id = threadIdx.x % warp_len; // id of a thread in a subwarp 
	int warp_num = tid / warp_len;
	// mask that is true for threads in the same subwarp
	unsigned same_threads = __match_any_sync(0xffffffff, warp_num);
	if (warp_id==0) shared_job[(warp_num&3)] = -1;

	/* Iterating over jobs/alignments */
	for (job_idx = job_start_idx; job_idx < job_end_idx; job_idx++) {
		
		/*Uneven Bucketing*/
		// the first subwarp fetches a long sequence's idx, while the remaining subwarps fetch short sequences' idx
		ub_idx = ((job_idx&3)==0)? global_ub_idx[n_tasks-(job_idx>>2)-1].y: global_ub_idx[job_idx-(job_idx>>2)-1].y;
		
		// get target and query sequence information
		packed_ref_batch_idx = target_batch_offsets[ub_idx] >> 3; //starting index of the target_batch sequence
		packed_query_batch_idx = query_batch_offsets[ub_idx] >> 3;//starting index of the query_batch sequence
		query_len = query_batch_lens[ub_idx]; // query sequence length
		ref_len = target_batch_lens[ub_idx]; // reference sequence length 
		packed_query_len = (query_len >> 3) + (query_len & 7 ? 1 : 0);//number of 32-bit words holding query_batch sequence
		packed_ref_len = (ref_len >> 3) + (ref_len & 7 ? 1 : 0);//number of 32-bit words holding target_batch sequence

		// check if alignment in the chain is done 

		/*Buffer Initialization*/
		// fill global buffer with initial value
		// global_buffer_top: used to store intermediate scores H and E in the horizontal strip (scores from the top)
		for (i = 0; i < job_per_query; i++) {
			l = i*warp_len + warp_id;
			if ((l) < max_query_len) {
				k = -(_cudaGapOE + (_cudaGapExtend*(l)));
				global_buffer_top[warp_num*max_query_len + l] =  l <= bw? make_short2(k, k-_cudaGapOE):initHD;	
			}
		}
		// global_buffer_left: used to store intermediate scores H and F in the vertical strip (scores from the left)
		for (i = 0; i < job_per_query; i++) {
			l = i*warp_len + warp_id;
			if ((l) < max_query_len) {
				k = -(_cudaGapOE + (_cudaGapExtend*(l)));
				global_buffer_left[warp_num*max_query_len + l] =  l <= bw? make_short2(k, k-_cudaGapOE):initHD;	
			}
		}
		// global_buffer_topleft: used to store intermediate scores H in the diagonal strip (scores from the top-left)
		for (i = 0; i < job_per_query; i++) {
			l = i*warp_len + warp_id;
			if (l < max_query_len) {
				k = -(_cudaGapOE+(_cudaGapExtend*(l*packed_len-1)));
				global_buffer_topleft[warp_num*max_query_len + l] = l==0? 0: (l*packed_len-1) <= bw? k: MINUS_INF2; 	
			}
		}
		// fill shared memory with initial value
		for (m = 0; m < total_shm; m++) {
			antidiag_max[real_warp_id + m*32] = INT_MIN;
		}

		__syncwarp();

		// Initialize variables
		max_score = 0; 
		prev_max_score = 0;
		max_ref_idx = 0; 
    	max_query_idx = 0;
		terminated = false;

		// check termination condition

		i = 0; //chunk
		total_anti_diags = packed_ref_len + packed_query_len-1; //chunk

		/*Subwarp Rejoining*/
		//set shared memory that is used to maintain values for subwarp rejoining
		if (warp_id==0) shared_job[(warp_num&3)] = total_anti_diags;
		else if (warp_id==1) shared_job[4+(warp_num&3)] = packed_ref_batch_idx;
		else if (warp_id==2) shared_job[8+(warp_num&3)] = packed_query_batch_idx;
		else if (warp_id==3) shared_job[12+(warp_num&3)] = (ref_len<<16)+query_len;
		else if (warp_id==4) shared_job[16+(warp_num&3)] = ub_idx;

		same_threads = __match_any_sync(__activemask(), warp_num);

		__syncwarp();

		/*Main Alignment Loop*/
		while (i < total_anti_diags) {
			
			// set boundaries for current slice
			slice_start = max(0, (i-packed_query_len+1));
			slice_start = max(slice_start, (i*packed_len + packed_len-1+1 - bw)/2/packed_len);
			slice_end = min(packed_ref_len-1, i+_cudaSliceWidth-1);
			slice_end = min(slice_end, ((i+_cudaSliceWidth-1)*packed_len + packed_len-1 + bw)/2/packed_len);
			finished_blocks = slice_start;
			
			if (slice_start > slice_end) {
				terminated = true;
			}

			while (!terminated && finished_blocks <= slice_end) {
				// while the entire chunk diag is not finished
				packed_ref_idx = finished_blocks + warp_id;
				packed_query_idx = i - packed_ref_idx;
				active = (packed_ref_idx <= slice_end);	//whether the current thread has cells to fill or not
				
				if (active) {
					ref_idx = packed_ref_idx << 3;
					query_idx = packed_query_idx << 3;

					// load intermediate values from global buffers
					p[1] = global_buffer_topleft[warp_num*max_query_len + packed_ref_idx];

					for (m = 1; m < 9; m++) {
						if ( (ref_idx + m-1) < ref_len) {
							HD = global_buffer_left[warp_num*max_query_len + ref_idx + m-1];
							h[m] = HD.x;
							f[m] = HD.y;
						} else {
							// if index out of bound of the score table 
							h[m] = MINUS_INF2;
							f[m] = MINUS_INF2;
						}
						
					}

					for (m=2;m<9;m++) {
						p[m] = h[m-1];
					}

					// Set boundaries for the current chunk
					chunk_start = (max(0, (packed_ref_idx*packed_len - bw)))/packed_len;
					chunk_end = min( packed_query_len-1, ( (packed_ref_idx*packed_len + packed_len -1 + bw)) /packed_len );
					packed_ref_literal = packed_ref_batch[packed_ref_batch_idx + packed_ref_idx];
				}
					
				// Compute the current chunk
				for (y = 0; y < _cudaSliceWidth; y++) {
					if (active && chunk_start <= packed_query_idx && packed_query_idx <= chunk_end) {
						packed_query_literal = packed_query_batch[packed_query_batch_idx + packed_query_idx]; 
						query_idx = packed_query_idx << 3;
						
						for (k = 28; k >= 0 && query_idx < query_len; k -= 4) {
							uint32_t qbase = (packed_query_literal >> k) & 15;	//get a base from query_batch sequence
							// load intermediate values from global buffers
							HD = global_buffer_top[warp_num*max_query_len + query_idx];
							h[0] = HD.x;
							e = HD.y;

							if (packed_query_idx == chunk_start || packed_query_idx == chunk_end) {
								#pragma unroll 8
								for (l = 28, m = 1; m < 9; l -= 4, m++) {
									CORE_COMPUTE_BOUNDARY();
								}
							} else {
								#pragma unroll 8
								for (l = 28, m = 1; m < 9; l -= 4, m++) {
									CORE_COMPUTE();
								}
							}
							
							// write intermediate values to global buffers
							HD.x = h[m-1];
							HD.y = e;
							global_buffer_top[warp_num*max_query_len + query_idx] = HD;

							//--------------- save dblock row -------------
							if ((ref_idx+m-1) % DBLOCK_SIZE == 0) {
								dblock_row[ub_idx * max_query_len * max_query_len / DBLOCK_SIZE + max_query_len * (ref_idx+m-1) / DBLOCK_SIZE + query_idx] = HD;
							}
							//---------------------------------------------

							query_idx++;

						}

					}
					

					packed_query_idx++;

					//-------------- save dblock col -------------------
					if ((packed_query_idx<<3)%DBLOCK_SIZE == 0 && active) { // TODO: merge with the loop below? idk
						short2 tmp_HD;
						for (int ridx = 0; ridx < 8; ridx++) {
							tmp_HD.x = h[ridx+1];
							tmp_HD.y = f[ridx+1];
							dblock_col[ub_idx * max_query_len * max_query_len / DBLOCK_SIZE 
							+ max_query_len * (packed_query_idx<<3) / DBLOCK_SIZE  
							+ ref_idx + ridx] = tmp_HD;
						}
					}
					//--------------------------------------------------
					
				}
				
				// write intermediate values to global buffers
				if (active) {	
					for (m = 1; m < 9; m++) {
						if ( ref_idx + m-1 < ref_len) {
							HD.x = h[m];
							HD.y = f[m];
							global_buffer_left[warp_num*max_query_len + ref_idx + m-1] = HD;
						}
					}
					global_buffer_topleft[warp_num*max_query_len + packed_ref_idx] = p[1];
				}
				
				finished_blocks+=warp_len;
			}

			__syncwarp();

			last_diag = (i+_cudaSliceWidth)<<3;
			prev_max_score = query_len+ref_len-1;

			/* Termination Condition & Score Update */
			if (!terminated) {
				for (diag_idx = i<<3; diag_idx < last_diag; diag_idx++) {
					if (diag_idx <prev_max_score) {
						m = diag_idx&(total_shm-1);
						temp = __reduce_max_sync(same_threads, antidiag_max[(m<<5)+real_warp_id]);
						if ((temp>>16) > max_score) {				
							max_score = temp>>16;
							max_ref_idx = (temp&65535);
							max_query_idx = diag_idx-max_ref_idx; 
						} else if ( (temp&65535) >= max_ref_idx && (diag_idx-(temp&65535)) >= max_query_idx) {
							int tl =  (temp&65535) - max_ref_idx, ql = (diag_idx-(temp&65535)) - max_query_idx, l;
							l = tl > ql? tl - ql : ql - tl;
							if (_cudaZThreshold >= 0 && max_score - (temp>>16) > _cudaZThreshold + l*_cudaGapExtend) {
								// Termination condition is met
								// write zdrop information to global drop array
								terminated = true;
								break;
							}
						}
						// reset shared memory buffer for next slice
						antidiag_max[(m<<5)+real_warp_id]=INT_MIN;
					}
				}
			}
			
			__syncwarp();

			// If job is finished
			if (terminated) {
				total_anti_diags = i; // set the total amount of diagonals as the current diagonal (to indicate that the job has finished)	
				if (warp_id==0) shared_job[(warp_num&3)] = total_anti_diags; //update this to shared memory as well (this will be used in Subwarp Rejoining as an indicator that the subwarp's job is done)
			}
			
			// Update the max score and its index to shared memory (used in Subwarp Rejoining)
			if (warp_id==1) shared_job[20+(warp_num&3)] = max_score;
			else if (warp_id==2) shared_job[24+(warp_num&3)] = (max_ref_idx<<16) + max_query_idx;
 
			__syncwarp();

			i += _cudaSliceWidth;

			/*Job wrap-up*/
			// If the job is done (either due to (1) meeting the termination condition (2) all the diagonals have been computed)
			if (i >= total_anti_diags) {
				
				// In the case of (2), check the termination condition & score update for the last diagonal block
				if (!terminated) {
					diag_idx = (i*packed_len)&(total_shm-1);
					for (k = i*packed_len, m = diag_idx; m < diag_idx+packed_len; m++, k++) {
						temp = __reduce_max_sync(same_threads, antidiag_max[(m<<5)+real_warp_id]);
						if ((temp>>16) > max_score) {				
							max_score = temp>>16;
							max_ref_idx = (temp&65535);
							max_query_idx = k-max_ref_idx; 
						} else if ( (temp&65535) >= max_ref_idx && (k-(temp&65535)) >= max_query_idx) {
							int tl =  (temp&65535) - max_ref_idx, ql = (k-(temp&65535)) - max_query_idx, l;
							l = tl > ql? tl - ql : ql - tl;
							if (_cudaZThreshold >= 0 && max_score - (temp>>16) > _cudaZThreshold + l*_cudaGapExtend) {
								// Termination condition is met
								terminated = true;
								break;
							}
						}
						antidiag_max[(m<<5)+real_warp_id]=INT_MIN;
					}
				}
				
				// Spill the results to GPU memory to be later moved to the CPU
				if (warp_id==0) {
					device_res->aln_score[ub_idx] = max_score;//copy the max score to the output array in the GPU mem
					device_res->query_batch_end[ub_idx] = max_query_idx;//copy the end position on query_batch sequence to the output array in the GPU mem
					device_res->target_batch_end[ub_idx] = max_ref_idx;//copy the end position on target_batch sequence to the output array in the GPU mem
				}

				/*Subwarp Rejoining*/
				// The subwarp that has no job looks for new jobs by iterating over other subwarp's job
				for (m = 0; m < (32/const_warp_len); m++) {
					// if the selected job still has remainig diagonals
					if (shared_job[m] > i) { // possible because all subwarps sync after each diagonal block is finished
						// read the selected job's info
						total_anti_diags = shared_job[m];
						warp_num = ((warp_num>>2)<<2)+m;
						ub_idx = shared_job[16+m];

						packed_ref_batch_idx = shared_job[4+m];
						packed_query_batch_idx = shared_job[8+m];
						ref_len = shared_job[12+m];
						query_len = ref_len&65535;
						ref_len = ref_len>>16;
						packed_query_len = (query_len >> 3) + (query_len & 7 ? 1 : 0);
						packed_ref_len = (ref_len >> 3) + (ref_len & 7 ? 1 : 0);
						
						max_score = shared_job[20+m];
						max_ref_idx = shared_job[24+m];
						max_query_idx = max_ref_idx&65535;
						max_ref_idx = max_ref_idx>>16;
						
						// reset the flag
						terminated = false;

						// reset shared memory buffer
						for (m = 0; m < total_shm; m++) {
							antidiag_max[(m<<5)+real_warp_id]=INT_MIN;
						}
						
						break;
					}
				}

			}

			__syncwarp();
			
			/*Subwarp Rejoining*/
			//Set the mask, warp length and thread id within the warp 
			same_threads = __match_any_sync(__activemask(), warp_num);
			warp_len = __popc(same_threads);
			warp_id = __popc((((0xffffffff) << (threadIdx.x % 32))&same_threads))-1;
			
			__syncwarp();

		}
		__syncwarp();
		/*Subwarp Rejoining*/
		//Reset subwarp and job related values for the next iteration
		warp_len = const_warp_len;
		warp_num = tid / warp_len;
		warp_id = tid % const_warp_len;
		ub_idx = shared_job[16+(warp_num&3)];

		__syncwarp();



	}
	
	return;


}


__global__ void agatha_kernel_approx_static_tb(uint32_t *packed_query_batch, uint32_t *packed_ref_batch,  uint32_t *query_batch_lens, uint32_t *target_batch_lens, uint32_t *query_batch_offsets, uint32_t *target_batch_offsets, gasal_res_t *device_res, gasal_res_t *device_res_second, uint32_t *packed_tb_matrices, int n_tasks, uint32_t max_query_len, short2 *global_buffer_top,
								bool* dropped, int bw)
{
    /*Initial kernel setup*/

	// Initializing variables 
	int32_t i, k, m, l, y, e;
	int32_t ub_idx, job_idx, ref_idx, query_idx;
	short2 HD;
	int32_t temp_score;
	int slice_start, slice_end, finished_blocks, chunk_start, chunk_end;
	int packed_ref_idx, packed_query_idx;
	int total_anti_diags;
	register uint32_t packed_ref_literal, packed_query_literal; 
	bool active, terminated;
	int32_t packed_ref_batch_idx, packed_query_batch_idx, query_len, ref_len, packed_query_len, packed_ref_len;
	int diag_idx, temp, last_diag;

	int test_id = 332;

	// Initializing max score and its idx
	__shared__ int32_t global_max_score[SHARED_SIZE]; 
	__shared__ int32_t global_max_ref_idx[SHARED_SIZE]; 
	__shared__ int32_t global_max_query_idx[SHARED_SIZE]; 
    __shared__ int32_t max_score[SHARED_SIZE]; 
	__shared__ int32_t max_ref_idx[SHARED_SIZE]; 
	__shared__ int32_t max_query_idx[SHARED_SIZE];
	bool max_block = true;

    int32_t prev_max_score = 0;

	// Setting constant values
	const short2 initHD = make_short2(MINUS_INF2, MINUS_INF2); //used to initialize short2
	const int32_t tid = (blockIdx.x * blockDim.x) + threadIdx.x; //thread ID within the entire kernel
	const int packed_len = 8; //number of bps (literals) packed into a single int32
	const int const_warp_len = 8; //number of threads per subwarp (before subwarp rejoining occurs)
	const int real_warp_id = threadIdx.x % 32; //thread ID within a single (full 32-thread) warp
	const int warp_per_kernel = (gridDim.x * blockDim.x) / const_warp_len; // number of subwarps. assume number of threads % const_warp_len == 0
	const int job_per_warp = n_tasks % warp_per_kernel ? (n_tasks / warp_per_kernel + 1) : n_tasks / warp_per_kernel; //number of jobs (alignments/tasks) needed to be done by a single subwarp
	const int job_per_query = max_query_len % const_warp_len ? (max_query_len / const_warp_len + 1) : max_query_len / const_warp_len; //number of a literal's initial score to fill per thread
	const int job_start_idx = (tid / const_warp_len)*job_per_warp; // the boundary of jobs of a subwarp 
	const int job_end_idx = (job_start_idx + job_per_warp) < n_tasks ? (job_start_idx + job_per_warp) : n_tasks; // the boundary of jobs of a subwarp
	const int total_shm = packed_len*(_cudaSliceWidth+1); // amount of shared memory a single thread uses
	
	// Arrays for saving intermediate values
	int32_t h[9];
	int32_t f[9];
	int32_t p[9];

	// Global memory setup
	short2* global_buffer_left = (short2*)(global_buffer_top+max_query_len*(blockDim.x/8)*gridDim.x);
	int32_t* global_buffer_topleft= (int32_t*)(global_buffer_left+max_query_len*(blockDim.x/8)*gridDim.x);
	short2* global_ub_idx = (short2*)(global_buffer_top+max_query_len*(blockDim.x/8)*gridDim.x*3);

	// Shared memory setup
	extern __shared__ int32_t shared_maxHH[];
	int32_t* shared_job = shared_maxHH+(blockDim.x/32)*total_shm*32+(threadIdx.x/32)*28;

	/* Setup values that will change after Subwarp Rejoining */
	int warp_len = const_warp_len;
	int warp_id = threadIdx.x % warp_len; // id of a thread in a subwarp 
	int warp_num = tid / warp_len;
	int warp_num_block = threadIdx.x / warp_len;
	// mask that is true for threads in the same subwarp
	unsigned same_threads = __match_any_sync(0xffffffff, warp_num);
	if (warp_id==0) shared_job[(warp_num&3)] = -1;

	/* Iterating over jobs/alignments */
	for (job_idx = job_start_idx; job_idx < job_end_idx; job_idx++) {
		
		/*Uneven Bucketing*/
		// the first subwarp fetches a long sequence's idx, while the remaining subwarps fetch short sequences' idx
		ub_idx = ((job_idx&3)==0)? global_ub_idx[n_tasks-(job_idx>>2)-1].y: global_ub_idx[job_idx-(job_idx>>2)-1].y;
		
		// get target and query sequence information
		packed_ref_batch_idx = target_batch_offsets[ub_idx] >> 3; //starting index of the target_batch sequence
		packed_query_batch_idx = query_batch_offsets[ub_idx] >> 3;//starting index of the query_batch sequence
		query_len = query_batch_lens[ub_idx]; // query sequence length
		ref_len = target_batch_lens[ub_idx]; // reference sequence length 
		packed_query_len = (query_len >> 3) + (query_len & 7 ? 1 : 0);//number of 32-bit words holding query_batch sequence
		packed_ref_len = (ref_len >> 3) + (ref_len & 7 ? 1 : 0);//number of 32-bit words holding target_batch sequence

		/*Buffer Initialization*/
		// fill global buffer with initial value
		// global_buffer_top: used to store intermediate scores H and E in the horizontal strip (scores from the top)
		for (i = 0; i < job_per_query; i++) {
			l = i*warp_len + warp_id;
			if ((l) < max_query_len) {
				k = -(_cudaGapOE + (_cudaGapExtend*(l)));
				global_buffer_top[warp_num*max_query_len + l] =  l <= bw? make_short2(k, k-_cudaGapOE):initHD;	
			}
		}
		// global_buffer_left: used to store intermediate scores H and F in the vertical strip (scores from the left)
		for (i = 0; i < job_per_query; i++) {
			l = i*warp_len + warp_id;
			if ((l) < max_query_len) {
				k = -(_cudaGapOE + (_cudaGapExtend*(l)));
				global_buffer_left[warp_num*max_query_len + l] =  l <= bw? make_short2(k, k-_cudaGapOE):initHD;	
			}
		}
		// global_buffer_topleft: used to store intermediate scores H in the diagonal strip (scores from the top-left)
		for (i = 0; i < job_per_query; i++) {
			l = i*warp_len + warp_id;
			if (l < max_query_len) {
				k = -(_cudaGapOE+(_cudaGapExtend*(l*packed_len-1)));
				global_buffer_topleft[warp_num*max_query_len + l] = l==0? 0: (l*packed_len-1) <= bw? k: MINUS_INF2; 	
			}
		}

		// Initialize variables
		global_max_score[warp_num_block] = 0;
		global_max_ref_idx[warp_num_block] = 0;
		global_max_query_idx[warp_num_block] = 0;
		max_score[warp_num_block] = 0; 
		prev_max_score = 0;
		max_ref_idx[warp_num_block] = 0; 
    	max_query_idx[warp_num_block] = 0;
		// terminated = false;

		// check termination condition
		terminated = dropped[ub_idx]; // TODO: fix this to be per-warp variable 

		__syncwarp();

		i = 0; //chunk
		total_anti_diags = packed_ref_len + packed_query_len-1; //chunk

		/*Subwarp Rejoining*/
		//set shared memory that is used to maintain values for subwarp rejoining
		if (warp_id==0) shared_job[(warp_num&3)] = total_anti_diags;
		else if (warp_id==1) shared_job[4+(warp_num&3)] = packed_ref_batch_idx;
		else if (warp_id==2) shared_job[8+(warp_num&3)] = packed_query_batch_idx;
		else if (warp_id==3) shared_job[12+(warp_num&3)] = (ref_len<<16)+query_len;
		else if (warp_id==4) shared_job[16+(warp_num&3)] = ub_idx;

		same_threads = __match_any_sync(__activemask(), warp_num);

		__syncwarp();

		/*Main Alignment Loop*/
		while (i < total_anti_diags) {
			
			// set boundaries for current slice
			slice_start = max(0, (i-packed_query_len+1));
			slice_start = max(slice_start, (i*packed_len + packed_len-1+1 - bw)/2/packed_len);
			slice_end = min(packed_ref_len-1, i+_cudaSliceWidth-1);
			slice_end = min(slice_end, ((i+_cudaSliceWidth-1)*packed_len + packed_len-1 + bw)/2/packed_len);
			finished_blocks = slice_start;
			
			if (slice_start > slice_end) {
				terminated = true;
			}

			while (!terminated && finished_blocks <= slice_end) {
				// while the entire chunk diag is not finished
				packed_ref_idx = finished_blocks + warp_id;
				packed_query_idx = i - packed_ref_idx;
				active = (packed_ref_idx <= slice_end);	//whether the current thread has cells to fill or not
				
				if (active) {
					ref_idx = packed_ref_idx << 3;
					query_idx = packed_query_idx << 3;

					// load intermediate values from global buffers
					p[1] = global_buffer_topleft[warp_num*max_query_len + packed_ref_idx];
			
					for (m = 1; m < 9; m++) {
						if ( (ref_idx + m-1) < ref_len) {
							HD = global_buffer_left[warp_num*max_query_len + ref_idx + m-1];
							h[m] = HD.x;
							f[m] = HD.y;
						} else {
							// if index out of bound of the score table 
							h[m] = MINUS_INF2;
							f[m] = MINUS_INF2;
						}
						
					}

					for (m=2;m<9;m++) {
						p[m] = h[m-1];
					}

					// Set boundaries for the current chunk
					chunk_start = (max(0, (packed_ref_idx*packed_len - bw)))/packed_len;
					chunk_end = min( packed_query_len-1, ( (packed_ref_idx*packed_len + packed_len -1 + bw)) /packed_len );
					packed_ref_literal = packed_ref_batch[packed_ref_batch_idx + packed_ref_idx];
				}
					
				// Compute the current chunk
				for (y = 0; y < _cudaSliceWidth; y++) {
					if (active && chunk_start <= packed_query_idx && packed_query_idx <= chunk_end) {
						packed_query_literal = packed_query_batch[packed_query_batch_idx + packed_query_idx]; 
						query_idx = packed_query_idx << 3;
						
						// set max index value if this thread has max cell in its 8*8 block
						if(((max_ref_idx[warp_num_block] >= ref_idx-1) && (max_ref_idx[warp_num_block] <= ref_idx+7)) || ((max_query_idx[warp_num_block] >= query_idx-1) && (max_query_idx[warp_num_block] <= query_idx+7)))
							max_block = true;
						else max_block = false;

						for (k = 28; k >= 0 && query_idx < query_len; k -= 4) {
							uint32_t qbase = (packed_query_literal >> k) & 15;	//get a base from query_batch sequence
							
							// load intermediate values from global buffers
							HD = global_buffer_top[warp_num*max_query_len + query_idx];
							h[0] = HD.x;
							e = HD.y;

							if (packed_query_idx == chunk_start || packed_query_idx == chunk_end) {
								if(max_block){
									#pragma unroll 8
									for (l = 28, m = 1; m < 9; l -= 4, m++) {
										CORE_COMPUTE_BOUNDARY_APPROX_MAX_TB(packed_tb_matrices[ub_idx*max_query_len*max_query_len/8 + packed_ref_idx * max_query_len + query_idx]);
									}
								}
								else{
									#pragma unroll 8
									for (l = 28, m = 1; m < 9; l -= 4, m++) {
										CORE_COMPUTE_BOUNDARY_APPROX_TB(packed_tb_matrices[ub_idx*max_query_len*max_query_len/8 + packed_ref_idx * max_query_len + query_idx]);
	
									}
								}
							} else if(max_block){
								#pragma unroll 8
								for (l = 28, m = 1; m < 9; l -= 4, m++) {
									CORE_COMPUTE_APPROX_MAX_TB(packed_tb_matrices[ub_idx*max_query_len*max_query_len/8 + packed_ref_idx * max_query_len + query_idx]);
								}
							} else{
								#pragma unroll 8
								for (l = 28, m = 1; m < 9; l -= 4, m++) {
									CORE_COMPUTE_APPROX_TB(packed_tb_matrices[ub_idx*max_query_len*max_query_len/8 + packed_ref_idx * max_query_len + query_idx]);
								}
							}
							

							// write intermediate values to global buffers
							HD.x = h[m-1];
							HD.y = e;
							global_buffer_top[warp_num*max_query_len + query_idx] = HD;

							query_idx++;

						}

					}
					

					packed_query_idx++;
					
				}
				
				// write intermediate values to global buffers
				if (active) {	
					for (m = 1; m < 9; m++) {
						if ( ref_idx + m-1 < ref_len) {
							HD.x = h[m];
							HD.y = f[m];
							global_buffer_left[warp_num*max_query_len + ref_idx + m-1] = HD;
						}
					}
					global_buffer_topleft[warp_num*max_query_len + packed_ref_idx] = p[1];
				}
				
				finished_blocks+=warp_len;
			}

			__syncwarp();

			last_diag = (i+_cudaSliceWidth)<<3;
			prev_max_score = query_len+ref_len-1;

			// check global max and z-drop condition for this slice
			
			__syncwarp();

			// If job is finished
			if (terminated) {
				total_anti_diags = i; // set the total amount of diagonals as the current diagonal (to indicate that the job has finished)	
				if (warp_id==0) shared_job[(warp_num&3)] = total_anti_diags; //update this to shared memory as well (this will be used in Subwarp Rejoining as an indicator that the subwarp's job is done)
			}
			
			// Update the max score and its index to shared memory (used in Subwarp Rejoining)
			if (warp_id==1) shared_job[20+(warp_num&3)] = warp_num_block;
			
			__syncwarp();

			i += _cudaSliceWidth;

			/*Job wrap-up*/
			// If the job is done (either due to (1) meeting the termination condition (2) all the diagonals have been computed)
			if (i >= total_anti_diags) {
				
				// Spill the results to GPU memory to be later moved to the CPU
				if (warp_id==0) {
					device_res->aln_score[ub_idx] = max_score[warp_num_block];//copy the max score to the output array in the GPU mem
					device_res->query_batch_end[ub_idx] = max_query_idx[warp_num_block];//copy the end position on query_batch sequence to the output array in the GPU mem
					device_res->target_batch_end[ub_idx] = max_ref_idx[warp_num_block];//copy the end position on target_batch sequence to the output array in the GPU mem
					dropped[ub_idx] = terminated;
				}

				/*Subwarp Rejoining*/
				// The subwarp that has no job looks for new jobs by iterating over other subwarp's job
				for (m = 0; m < (32/const_warp_len); m++) {
					// if the selected job still has remainig diagonals
					if (shared_job[m] > i) { // possible because all subwarps sync after each diagonal block is finished
						// read the selected job's info
						total_anti_diags = shared_job[m];
						warp_num = ((warp_num>>2)<<2)+m;
						ub_idx = shared_job[16+m];

						packed_ref_batch_idx = shared_job[4+m];
						packed_query_batch_idx = shared_job[8+m];
						ref_len = shared_job[12+m];
						query_len = ref_len&65535;
						ref_len = ref_len>>16;
						packed_query_len = (query_len >> 3) + (query_len & 7 ? 1 : 0);
						packed_ref_len = (ref_len >> 3) + (ref_len & 7 ? 1 : 0);
						
						warp_num_block = shared_job[20+m];
						
						// reset the flag
						terminated = dropped[ub_idx];
						
						break;
					}
				}

			}

			__syncwarp();
			
			/*Subwarp Rejoining*/
			//Set the mask, warp length and thread id within the warp 
			same_threads = __match_any_sync(__activemask(), warp_num);
			warp_len = __popc(same_threads);
			warp_id = __popc((((0xffffffff) << (threadIdx.x % 32))&same_threads))-1;
			
			__syncwarp();

		}
		__syncwarp();
		/*Subwarp Rejoining*/
		//Reset subwarp and job related values for the next iteration
		warp_len = const_warp_len;
		warp_num = tid / warp_len;
		warp_id = tid % const_warp_len;
		ub_idx = shared_job[16+(warp_num&3)];

		__syncwarp();



	}
	
	return;


}

__global__ void agatha_kernel_approx_dynamic_tb(uint32_t *packed_query_batch, uint32_t *packed_ref_batch,  uint32_t *query_batch_lens, uint32_t *target_batch_lens, uint32_t *query_batch_offsets, uint32_t *target_batch_offsets, gasal_res_t *device_res, gasal_res_t *device_res_second, short2 *dblock_row, short2 *dblock_col, int n_tasks, uint32_t max_query_len, short2 *global_buffer_top,
								bool* dropped, int bw)
{
    /*Initial kernel setup*/

	// Initializing variables 
	int32_t i, k, m, l, y, e;
	int32_t ub_idx, job_idx, ref_idx, query_idx;
	short2 HD;
	int32_t temp_score;
	int slice_start, slice_end, finished_blocks, chunk_start, chunk_end;
	int packed_ref_idx, packed_query_idx;
	int total_anti_diags;
	register uint32_t packed_ref_literal, packed_query_literal; 
	bool active, terminated;
	int32_t packed_ref_batch_idx, packed_query_batch_idx, query_len, ref_len, packed_query_len, packed_ref_len;
	int diag_idx, temp, last_diag;

	// Initializing max score and its idx
	__shared__ int32_t global_max_score[SHARED_SIZE]; 
	__shared__ int32_t global_max_ref_idx[SHARED_SIZE]; 
	__shared__ int32_t global_max_query_idx[SHARED_SIZE]; 
    __shared__ int32_t max_score[SHARED_SIZE]; 
	__shared__ int32_t max_ref_idx[SHARED_SIZE]; 
	__shared__ int32_t max_query_idx[SHARED_SIZE];
	bool max_block = true;

    int32_t prev_max_score = 0;

	// Setting constant values
	const short2 initHD = make_short2(MINUS_INF2, MINUS_INF2); //used to initialize short2
	const int32_t tid = (blockIdx.x * blockDim.x) + threadIdx.x; //thread ID within the entire kernel
	const int packed_len = 8; //number of bps (literals) packed into a single int32
	const int const_warp_len = 8; //number of threads per subwarp (before subwarp rejoining occurs)
	const int real_warp_id = threadIdx.x % 32; //thread ID within a single (full 32-thread) warp
	const int warp_per_kernel = (gridDim.x * blockDim.x) / const_warp_len; // number of subwarps. assume number of threads % const_warp_len == 0
	const int job_per_warp = n_tasks % warp_per_kernel ? (n_tasks / warp_per_kernel + 1) : n_tasks / warp_per_kernel; //number of jobs (alignments/tasks) needed to be done by a single subwarp
	const int job_per_query = max_query_len % const_warp_len ? (max_query_len / const_warp_len + 1) : max_query_len / const_warp_len; //number of a literal's initial score to fill per thread
	const int job_start_idx = (tid / const_warp_len)*job_per_warp; // the boundary of jobs of a subwarp 
	const int job_end_idx = (job_start_idx + job_per_warp) < n_tasks ? (job_start_idx + job_per_warp) : n_tasks; // the boundary of jobs of a subwarp
	const int total_shm = packed_len*(_cudaSliceWidth+1); // amount of shared memory a single thread uses
	
	// Arrays for saving intermediate values
	int32_t h[9];
	int32_t f[9];
	int32_t p[9];

	// Global memory setup
	short2* global_buffer_left = (short2*)(global_buffer_top+max_query_len*(blockDim.x/8)*gridDim.x);
	int32_t* global_buffer_topleft= (int32_t*)(global_buffer_left+max_query_len*(blockDim.x/8)*gridDim.x);
	short2* global_ub_idx = (short2*)(global_buffer_top+max_query_len*(blockDim.x/8)*gridDim.x*3);

	// Shared memory setup
	extern __shared__ int32_t shared_maxHH[];
	int32_t* shared_job = shared_maxHH+(blockDim.x/32)*total_shm*32+(threadIdx.x/32)*28;

	/* Setup values that will change after Subwarp Rejoining */
	int warp_len = const_warp_len;
	int warp_id = threadIdx.x % warp_len; // id of a thread in a subwarp 
	int warp_num = tid / warp_len;
	int warp_num_block = threadIdx.x / warp_len;
	// mask that is true for threads in the same subwarp
	unsigned same_threads = __match_any_sync(0xffffffff, warp_num);
	if (warp_id==0) shared_job[(warp_num&3)] = -1;

	/* Iterating over jobs/alignments */
	for (job_idx = job_start_idx; job_idx < job_end_idx; job_idx++) {
		
		/*Uneven Bucketing*/
		// the first subwarp fetches a long sequence's idx, while the remaining subwarps fetch short sequences' idx
		ub_idx = ((job_idx&3)==0)? global_ub_idx[n_tasks-(job_idx>>2)-1].y: global_ub_idx[job_idx-(job_idx>>2)-1].y;
		
		// get target and query sequence information
		packed_ref_batch_idx = target_batch_offsets[ub_idx] >> 3; //starting index of the target_batch sequence
		packed_query_batch_idx = query_batch_offsets[ub_idx] >> 3;//starting index of the query_batch sequence
		query_len = query_batch_lens[ub_idx]; // query sequence length
		ref_len = target_batch_lens[ub_idx]; // reference sequence length 
		packed_query_len = (query_len >> 3) + (query_len & 7 ? 1 : 0);//number of 32-bit words holding query_batch sequence
		packed_ref_len = (ref_len >> 3) + (ref_len & 7 ? 1 : 0);//number of 32-bit words holding target_batch sequence

		// check if alignment in the chain is done 
	
		/*Buffer Initialization*/
		// fill global buffer with initial value
		// global_buffer_top: used to store intermediate scores H and E in the horizontal strip (scores from the top)
		for (i = 0; i < job_per_query; i++) {
			l = i*warp_len + warp_id;
			if ((l) < max_query_len) {
				k = -(_cudaGapOE + (_cudaGapExtend*(l)));
				global_buffer_top[warp_num*max_query_len + l] =  l <= bw? make_short2(k, k-_cudaGapOE):initHD;	
			}
		}
		// global_buffer_left: used to store intermediate scores H and F in the vertical strip (scores from the left)
		for (i = 0; i < job_per_query; i++) {
			l = i*warp_len + warp_id;
			if ((l) < max_query_len) {
				k = -(_cudaGapOE + (_cudaGapExtend*(l)));
				global_buffer_left[warp_num*max_query_len + l] =  l <= bw? make_short2(k, k-_cudaGapOE):initHD;	
			}
		}
		// global_buffer_topleft: used to store intermediate scores H in the diagonal strip (scores from the top-left)
		for (i = 0; i < job_per_query; i++) {
			l = i*warp_len + warp_id;
			if (l < max_query_len) {
				k = -(_cudaGapOE+(_cudaGapExtend*(l*packed_len-1)));
				global_buffer_topleft[warp_num*max_query_len + l] = l==0? 0: (l*packed_len-1) <= bw? k: MINUS_INF2; 	
			}
		}

		// Initialize variables
		global_max_score[warp_num_block] = 0;
		global_max_ref_idx[warp_num_block] = 0;
		global_max_query_idx[warp_num_block] = 0;
		max_score[warp_num_block] = 0; 
		prev_max_score = 0;
		max_ref_idx[warp_num_block] = 0; 
    	max_query_idx[warp_num_block] = 0;
		// terminated = false;

		// check termination condition
		terminated = dropped[ub_idx]; // TODO: fix this to be per-warp variable 

		__syncwarp();

		i = 0; //chunk
		total_anti_diags = packed_ref_len + packed_query_len-1; //chunk

		/*Subwarp Rejoining*/
		//set shared memory that is used to maintain values for subwarp rejoining
		if (warp_id==0) shared_job[(warp_num&3)] = total_anti_diags;
		else if (warp_id==1) shared_job[4+(warp_num&3)] = packed_ref_batch_idx;
		else if (warp_id==2) shared_job[8+(warp_num&3)] = packed_query_batch_idx;
		else if (warp_id==3) shared_job[12+(warp_num&3)] = (ref_len<<16)+query_len;
		else if (warp_id==4) shared_job[16+(warp_num&3)] = ub_idx;

		same_threads = __match_any_sync(__activemask(), warp_num);

		__syncwarp();

		/*Main Alignment Loop*/
		while (i < total_anti_diags) {
			
			// set boundaries for current slice
			slice_start = max(0, (i-packed_query_len+1));
			slice_start = max(slice_start, (i*packed_len + packed_len-1+1 - bw)/2/packed_len);
			slice_end = min(packed_ref_len-1, i+_cudaSliceWidth-1);
			slice_end = min(slice_end, ((i+_cudaSliceWidth-1)*packed_len + packed_len-1 + bw)/2/packed_len);
			finished_blocks = slice_start;
			
			if (slice_start > slice_end) {
				terminated = true;
			}

			while (!terminated && finished_blocks <= slice_end) {
				// while the entire chunk diag is not finished
				packed_ref_idx = finished_blocks + warp_id;
				packed_query_idx = i - packed_ref_idx;
				active = (packed_ref_idx <= slice_end);	//whether the current thread has cells to fill or not
				
				if (active) {
					ref_idx = packed_ref_idx << 3;
					query_idx = packed_query_idx << 3;

					// load intermediate values from global buffers
					p[1] = global_buffer_topleft[warp_num*max_query_len + packed_ref_idx];
			
					for (m = 1; m < 9; m++) {
						if ( (ref_idx + m-1) < ref_len) {
							HD = global_buffer_left[warp_num*max_query_len + ref_idx + m-1];
							h[m] = HD.x;
							f[m] = HD.y;
						} else {
							// if index out of bound of the score table 
							h[m] = MINUS_INF2;
							f[m] = MINUS_INF2;
						}
						
					}

					for (m=2;m<9;m++) {
						p[m] = h[m-1];
					}

					// Set boundaries for the current chunk
					chunk_start = (max(0, (packed_ref_idx*packed_len - bw)))/packed_len;
					chunk_end = min( packed_query_len-1, ( (packed_ref_idx*packed_len + packed_len -1 + bw)) /packed_len );
					packed_ref_literal = packed_ref_batch[packed_ref_batch_idx + packed_ref_idx];
				}
					
				// Compute the current chunk
				for (y = 0; y < _cudaSliceWidth; y++) {
					if (active && chunk_start <= packed_query_idx && packed_query_idx <= chunk_end) {
						packed_query_literal = packed_query_batch[packed_query_batch_idx + packed_query_idx]; 
						query_idx = packed_query_idx << 3;

					
						// set max index value if this thread has max cell in its 8*8 block
						if(((max_ref_idx[warp_num_block] >= ref_idx-1) && (max_ref_idx[warp_num_block] <= ref_idx+7)) || ((max_query_idx[warp_num_block] >= query_idx-1) && (max_query_idx[warp_num_block] <= query_idx+7)))
							max_block = true;
						else max_block = false;

						for (k = 28; k >= 0 && query_idx < query_len; k -= 4) {
							uint32_t qbase = (packed_query_literal >> k) & 15;	//get a base from query_batch sequence
							
							// load intermediate values from global buffers
							HD = global_buffer_top[warp_num*max_query_len + query_idx];
							h[0] = HD.x;
							e = HD.y;

							if (packed_query_idx == chunk_start || packed_query_idx == chunk_end) {
								if(max_block){
									#pragma unroll 8
									for (l = 28, m = 1; m < 9; l -= 4, m++) {
										CORE_COMPUTE_BOUNDARY_APPROX_MAX();
									}
								}
								else{
									#pragma unroll 8
									for (l = 28, m = 1; m < 9; l -= 4, m++) {
										CORE_COMPUTE_BOUNDARY_APPROX();
	
									}
								}
							} else if(max_block){
								#pragma unroll 8
								for (l = 28, m = 1; m < 9; l -= 4, m++) {
									CORE_COMPUTE_APPROX_MAX();
								}
							} else{
								#pragma unroll 8
								for (l = 28, m = 1; m < 9; l -= 4, m++) {
									CORE_COMPUTE_APPROX();
								}
							}
							
							// write intermediate values to global buffers
							HD.x = h[m-1];
							HD.y = e;
							global_buffer_top[warp_num*max_query_len + query_idx] = HD;

							//--------------- save dblock row -------------
							if ((ref_idx+m-1) % DBLOCK_SIZE == 0) {
								size_t dblock_row_ofs = (size_t)ub_idx * (size_t)max_query_len * (size_t)max_query_len / DBLOCK_SIZE + (size_t)max_query_len * (size_t)(ref_idx+m-1) / DBLOCK_SIZE + (size_t)query_idx;
								dblock_row[dblock_row_ofs] = HD;
							}
							//---------------------------------------------
							query_idx++;
							
						}

				

						//-------------- save dblock col -------------------
						if (((packed_query_idx+1)<<3)%DBLOCK_SIZE == 0 && packed_query_idx > 0) { // TODO: merge with the loop below? idk
							short2 tmp_HD;
							for (int ridx = 0; ridx < 8; ridx++) {
								tmp_HD.x = h[ridx+1];
								tmp_HD.y = f[ridx+1];
								size_t dblock_col_ofs = (size_t)ub_idx * (size_t)max_query_len * (size_t)max_query_len / DBLOCK_SIZE 
								+ (size_t)max_query_len * (size_t)((packed_query_idx+1)<<3) / DBLOCK_SIZE  
								+ (size_t)ref_idx + (size_t)ridx;
								dblock_col[dblock_col_ofs] = tmp_HD;
							}
						}
						//--------------------------------------------------


					}
					
					packed_query_idx++;
					
				}
				
				// write intermediate values to global buffers
				if (active) {	
					for (m = 1; m < 9; m++) {
						if ( ref_idx + m-1 < ref_len) {
							HD.x = h[m];
							HD.y = f[m];
							global_buffer_left[warp_num*max_query_len + ref_idx + m-1] = HD;
						}
					}
					global_buffer_topleft[warp_num*max_query_len + packed_ref_idx] = p[1];
				}
				
				finished_blocks+=warp_len;
			}

			__syncwarp();

			last_diag = (i+_cudaSliceWidth)<<3;
			prev_max_score = query_len+ref_len-1;
			
			__syncwarp();

			// If job is finished
			if (terminated) {
				total_anti_diags = i; // set the total amount of diagonals as the current diagonal (to indicate that the job has finished)	
				if (warp_id==0) shared_job[(warp_num&3)] = total_anti_diags; //update this to shared memory as well (this will be used in Subwarp Rejoining as an indicator that the subwarp's job is done)
			}
			
			// Update the max score and its index to shared memory (used in Subwarp Rejoining)
			if (warp_id==1) shared_job[20+(warp_num&3)] = warp_num_block;
		
			__syncwarp();

			i += _cudaSliceWidth;

			/*Job wrap-up*/
			// If the job is done (either due to (1) meeting the termination condition (2) all the diagonals have been computed)
			if (i >= total_anti_diags) {
				
				// Spill the results to GPU memory to be later moved to the CPU
				if (warp_id==0) {
					/* TODO: fix warp num block issue */
					if(warp_num_block < SHARED_SIZE && warp_num_block >= 0) {
						device_res->aln_score[ub_idx] = global_max_score[warp_num_block];//copy the max score to the output array in the GPU mem
						device_res->query_batch_end[ub_idx] = global_max_query_idx[warp_num_block];//copy the end position on query_batch sequence to the output array in the GPU mem
						device_res->target_batch_end[ub_idx] = global_max_ref_idx[warp_num_block];//copy the end position on target_batch sequence to the output array in the GPU mem
					}
			
					dropped[ub_idx] = terminated;
				}

				/*Subwarp Rejoining*/
				// The subwarp that has no job looks for new jobs by iterating over other subwarp's job
				for (m = 0; m < (32/const_warp_len); m++) {
					// if the selected job still has remainig diagonals
					if (shared_job[m] > i) { // possible because all subwarps sync after each diagonal block is finished
						// read the selected job's info
						total_anti_diags = shared_job[m];
						warp_num = ((warp_num>>2)<<2)+m;
						ub_idx = shared_job[16+m];

						packed_ref_batch_idx = shared_job[4+m];
						packed_query_batch_idx = shared_job[8+m];
						ref_len = shared_job[12+m];
						query_len = ref_len&65535;
						ref_len = ref_len>>16;
						packed_query_len = (query_len >> 3) + (query_len & 7 ? 1 : 0);
						packed_ref_len = (ref_len >> 3) + (ref_len & 7 ? 1 : 0);
						
						warp_num_block = shared_job[20+m];
					
						// reset the flag
						terminated = dropped[ub_idx];
						
						break;
					}
				}

			}

			__syncwarp();
			
			/*Subwarp Rejoining*/
			//Set the mask, warp length and thread id within the warp 
			same_threads = __match_any_sync(__activemask(), warp_num);
			warp_len = __popc(same_threads);
			warp_id = __popc((((0xffffffff) << (threadIdx.x % 32))&same_threads))-1;
			
			__syncwarp();

		}
		__syncwarp();
		/*Subwarp Rejoining*/
		//Reset subwarp and job related values for the next iteration
		warp_len = const_warp_len;
		warp_num = tid / warp_len;
		warp_id = tid % const_warp_len;
		ub_idx = shared_job[16+(warp_num&3)];

		__syncwarp();

	}
	return;


}


__global__ void agatha_kernel_approx_dynamic_tb_offset(uint32_t *packed_query_batch, uint32_t *packed_ref_batch,  uint32_t *query_batch_lens, uint32_t *target_batch_lens, uint32_t *query_batch_offsets, uint32_t *target_batch_offsets, gasal_res_t *device_res, gasal_res_t *device_res_second, short2 *dblock_row, short2 *dblock_col, int n_tasks, uint32_t max_query_len, short2 *global_buffer_top,
	uint64_t* tb_ofs, bool* dropped, int bw)
{
/*Initial kernel setup*/

	// Initializing variables 
	int32_t i, k, m, l, y, e;
	int32_t ub_idx, job_idx, ref_idx, query_idx;
	short2 HD;
	int32_t temp_score;
	int slice_start, slice_end, finished_blocks, chunk_start, chunk_end;
	int packed_ref_idx, packed_query_idx;
	int total_anti_diags;
	register uint32_t packed_ref_literal, packed_query_literal; 
	bool active, terminated;
	int32_t packed_ref_batch_idx, packed_query_batch_idx, query_len, ref_len, packed_query_len, packed_ref_len;
	int diag_idx, temp, last_diag;

	uint64_t tb_offset;

	// Initializing max score and its idx
	__shared__ int32_t global_max_score[SHARED_SIZE]; 
	__shared__ int32_t global_max_ref_idx[SHARED_SIZE]; 
	__shared__ int32_t global_max_query_idx[SHARED_SIZE]; 
	__shared__ int32_t max_score[SHARED_SIZE]; 
	__shared__ int32_t max_ref_idx[SHARED_SIZE]; 
	__shared__ int32_t max_query_idx[SHARED_SIZE];
	bool max_block = true;

	int32_t prev_max_score = 0;
	
	// Setting constant values
	const short2 initHD = make_short2(MINUS_INF2, MINUS_INF2); //used to initialize short2
	const int32_t tid = (blockIdx.x * blockDim.x) + threadIdx.x; //thread ID within the entire kernel
	const int packed_len = 8; //number of bps (literals) packed into a single int32
	const int const_warp_len = 8; //number of threads per subwarp (before subwarp rejoining occurs)
	const int real_warp_id = threadIdx.x % 32; //thread ID within a single (full 32-thread) warp
	const int warp_per_kernel = (gridDim.x * blockDim.x) / const_warp_len; // number of subwarps. assume number of threads % const_warp_len == 0
	const int job_per_warp = n_tasks % warp_per_kernel ? (n_tasks / warp_per_kernel + 1) : n_tasks / warp_per_kernel; //number of jobs (alignments/tasks) needed to be done by a single subwarp
	const int job_per_query = max_query_len % const_warp_len ? (max_query_len / const_warp_len + 1) : max_query_len / const_warp_len; //number of a literal's initial score to fill per thread
	const int job_start_idx = (tid / const_warp_len)*job_per_warp; // the boundary of jobs of a subwarp 
	const int job_end_idx = (job_start_idx + job_per_warp) < n_tasks ? (job_start_idx + job_per_warp) : n_tasks; // the boundary of jobs of a subwarp
	const int total_shm = packed_len*(_cudaSliceWidth+1); // amount of shared memory a single thread uses

	// Arrays for saving intermediate values
	int32_t h[9];
	int32_t f[9];
	int32_t p[9];

	// Global memory setup
	short2* global_buffer_left = (short2*)(global_buffer_top+max_query_len*(blockDim.x/8)*gridDim.x);
	int32_t* global_buffer_topleft= (int32_t*)(global_buffer_left+max_query_len*(blockDim.x/8)*gridDim.x);
	short2* global_ub_idx = (short2*)(global_buffer_top+max_query_len*(blockDim.x/8)*gridDim.x*3);

	// Shared memory setup
	extern __shared__ int32_t shared_maxHH[];
	int32_t* shared_job = shared_maxHH+(blockDim.x/32)*total_shm*32+(threadIdx.x/32)*28;

	/* Setup values that will change after Subwarp Rejoining */
	int warp_len = const_warp_len;
	int warp_id = threadIdx.x % warp_len; // id of a thread in a subwarp 
	int warp_num = tid / warp_len;
	int warp_num_block = threadIdx.x / warp_len;
	// mask that is true for threads in the same subwarp
	unsigned same_threads = __match_any_sync(0xffffffff, warp_num);
	if (warp_id==0) shared_job[(warp_num&3)] = -1;

	bool test = warp_num < 4 && warp_id==0; // test for first warp..

	/* Iterating over jobs/alignments */
	for (job_idx = job_start_idx; job_idx < job_end_idx; job_idx++) {

		/*Uneven Bucketing*/
		// the first subwarp fetches a long sequence's idx, while the remaining subwarps fetch short sequences' idx
		ub_idx = ((job_idx&3)==0)? static_cast<uint16_t>(global_ub_idx[n_tasks-(job_idx>>2)-1].y): static_cast<uint16_t>(global_ub_idx[job_idx-(job_idx>>2)-1].y);

		// get target and query sequence information
		packed_ref_batch_idx = target_batch_offsets[ub_idx] >> 3; //starting index of the target_batch sequence
		packed_query_batch_idx = query_batch_offsets[ub_idx] >> 3;//starting index of the query_batch sequence
		query_len = query_batch_lens[ub_idx]; // query sequence length
		ref_len = target_batch_lens[ub_idx]; // reference sequence length 
		packed_query_len = (query_len >> 3) + (query_len & 7 ? 1 : 0);//number of 32-bit words holding query_batch sequence
		packed_ref_len = (ref_len >> 3) + (ref_len & 7 ? 1 : 0);//number of 32-bit words holding target_batch sequence

		// check if alignment in the chain is done 
		terminated = dropped[ub_idx];
		
		tb_offset = tb_ofs[ub_idx]; // TODO: check if this if correct

		/*Buffer Initialization*/
		// fill global buffer with initial value
		// global_buffer_top: used to store intermediate scores H and E in the horizontal strip (scores from the top)
		for (i = 0; i < job_per_query; i++) {
			l = i*warp_len + warp_id;
			if ((l) < max_query_len) {
				k = -(_cudaGapOE + (_cudaGapExtend*(l)));
				global_buffer_top[warp_num*max_query_len + l] =  l <= bw? make_short2(k, k-_cudaGapOE):initHD;	
			}
		}
		// global_buffer_left: used to store intermediate scores H and F in the vertical strip (scores from the left)
		for (i = 0; i < job_per_query; i++) {
			l = i*warp_len + warp_id;
			if ((l) < max_query_len) {
				k = -(_cudaGapOE + (_cudaGapExtend*(l)));
				global_buffer_left[warp_num*max_query_len + l] =  l <= bw? make_short2(k, k-_cudaGapOE):initHD;	
			}
		}
		// global_buffer_topleft: used to store intermediate scores H in the diagonal strip (scores from the top-left)
		for (i = 0; i < job_per_query; i++) {
			l = i*warp_len + warp_id;
			if (l < max_query_len) {
				k = -(_cudaGapOE+(_cudaGapExtend*(l*packed_len-1)));
				global_buffer_topleft[warp_num*max_query_len + l] = l==0? 0: (l*packed_len-1) <= bw? k: MINUS_INF2; 	
			}
		}

		// Initialize variables
		global_max_score[warp_num_block] = 0;
		global_max_ref_idx[warp_num_block] = 0;
		global_max_query_idx[warp_num_block] = 0;
		max_score[warp_num_block] = 0; 
		prev_max_score = 0;
		max_ref_idx[warp_num_block] = 0; 
		max_query_idx[warp_num_block] = 0;
		
		__syncwarp();

		i = 0; //chunk
		total_anti_diags = packed_ref_len + packed_query_len-1; //chunk
		
		/*Subwarp Rejoining*/
		//set shared memory that is used to maintain values for subwarp rejoining
		if (warp_id==0) shared_job[(warp_num&3)] = total_anti_diags;
		else if (warp_id==1) shared_job[4+(warp_num&3)] = packed_ref_batch_idx;
		else if (warp_id==2) shared_job[8+(warp_num&3)] = packed_query_batch_idx;
		else if (warp_id==3) shared_job[12+(warp_num&3)] = (ref_len<<16)+query_len;
		else if (warp_id==4) shared_job[16+(warp_num&3)] = ub_idx;

		same_threads = __match_any_sync(__activemask(), warp_num);

		__syncwarp();

		/*Main Alignment Loop*/
		while (i < total_anti_diags) {
			//if(test) printf("%d\t[%d] %d / %d %d warp num %d \n", ub_idx, warp_id, i, total_anti_diags, terminated, warp_num);

			// set boundaries for current slice
			slice_start = max(0, (i-packed_query_len+1));
			slice_start = max(slice_start, (i*packed_len + packed_len-1+1 - bw)/2/packed_len);
			slice_end = min(packed_ref_len-1, i+_cudaSliceWidth-1);
			slice_end = min(slice_end, ((i+_cudaSliceWidth-1)*packed_len + packed_len-1 + bw)/2/packed_len);
			finished_blocks = slice_start;

			if (slice_start > slice_end) {
			terminated = true;
			}

			while (!terminated && finished_blocks <= slice_end) {
				// while the entire chunk diag is not finished
				packed_ref_idx = finished_blocks + warp_id;
				packed_query_idx = i - packed_ref_idx;
				active = (packed_ref_idx <= slice_end);	//whether the current thread has cells to fill or not

				if (active) {
					ref_idx = packed_ref_idx << 3;
					query_idx = packed_query_idx << 3;

					// load intermediate values from global buffers
					p[1] = global_buffer_topleft[warp_num*max_query_len + packed_ref_idx];

					for (m = 1; m < 9; m++) {
						if ( (ref_idx + m-1) < ref_len) {
							HD = global_buffer_left[warp_num*max_query_len + ref_idx + m-1];
							h[m] = HD.x;
							f[m] = HD.y;
						} else {
							// if index out of bound of the score table 
							h[m] = MINUS_INF2;
							f[m] = MINUS_INF2;
						}

					}

					for (m=2;m<9;m++) {
						p[m] = h[m-1];
					}

					// Set boundaries for the current chunk
					chunk_start = (max(0, (packed_ref_idx*packed_len - bw)))/packed_len;
					chunk_end = min( packed_query_len-1, ( (packed_ref_idx*packed_len + packed_len -1 + bw)) /packed_len );
					packed_ref_literal = packed_ref_batch[packed_ref_batch_idx + packed_ref_idx];
				}

				// Compute the current chunk
				for (y = 0; y < _cudaSliceWidth; y++) {
					if (active && chunk_start <= packed_query_idx && packed_query_idx <= chunk_end) {
						packed_query_literal = packed_query_batch[packed_query_batch_idx + packed_query_idx]; 
						query_idx = packed_query_idx << 3;

						// set max index value if this thread has max cell in its 8*8 block
						if(((max_ref_idx[warp_num_block] >= ref_idx-1) && (max_ref_idx[warp_num_block] <= ref_idx+7)) || ((max_query_idx[warp_num_block] >= query_idx-1) && (max_query_idx[warp_num_block] <= query_idx+7)))
						max_block = true;
						else max_block = false;

						for (k = 28; k >= 0 && query_idx < query_len; k -= 4) {
							uint32_t qbase = (packed_query_literal >> k) & 15;	//get a base from query_batch sequence

							// load intermediate values from global buffers
							HD = global_buffer_top[warp_num*max_query_len + query_idx];
							h[0] = HD.x;
							e = HD.y;

							if (packed_query_idx == chunk_start || packed_query_idx == chunk_end) {
								if(max_block){
									#pragma unroll 8
									for (l = 28, m = 1; m < 9; l -= 4, m++) {
										CORE_COMPUTE_BOUNDARY_APPROX_MAX();
									}
								}	
								else{
									#pragma unroll 8
									for (l = 28, m = 1; m < 9; l -= 4, m++) {
										CORE_COMPUTE_BOUNDARY_APPROX();
									}
								}
							} else if(max_block){
								#pragma unroll 8
								for (l = 28, m = 1; m < 9; l -= 4, m++) {
									CORE_COMPUTE_APPROX_MAX();
								}
							} else{
								#pragma unroll 8
								for (l = 28, m = 1; m < 9; l -= 4, m++) {
								CORE_COMPUTE_APPROX();
								}
							}

							// write intermediate values to global buffers
							HD.x = h[m-1];
							HD.y = e;
							global_buffer_top[warp_num*max_query_len + query_idx] = HD;

							//--------------- save dblock row -------------
							if ((ref_idx+m-1) % DBLOCK_SIZE_D == 0) {
							size_t dblock_row_ofs = (size_t)tb_offset + (size_t)query_len * (size_t)((ref_idx+m-1) / DBLOCK_SIZE_D) + (size_t)query_idx;

							dblock_row[dblock_row_ofs] = HD;
						}
						//---------------------------------------------
						query_idx++;
					}

					//-------------- save dblock col -------------------
					if (((packed_query_idx+1)<<3)%DBLOCK_SIZE_D == 0) { 
						short2 tmp_HD;
						for (int ridx = 0; ridx < 8; ridx++) {
							tmp_HD.x = h[ridx+1];
							tmp_HD.y = f[ridx+1];
							
							size_t dblock_col_ofs = (size_t)tb_offset  
							+ (size_t)ref_len * (size_t)((packed_query_idx+1)<<3) / DBLOCK_SIZE_D  
							+ (size_t)ref_idx + (size_t)ridx;

							dblock_col[dblock_col_ofs] = tmp_HD;
						}
					}
				//--------------------------------------------------
				}

				packed_query_idx++;

			}

			// write intermediate values to global buffers
			if (active) {	
				for (m = 1; m < 9; m++) {
					if ( ref_idx + m-1 < ref_len) {
						HD.x = h[m];
						HD.y = f[m];
						global_buffer_left[warp_num*max_query_len + ref_idx + m-1] = HD;
					}
				}
				global_buffer_topleft[warp_num*max_query_len + packed_ref_idx] = p[1];
			}

			finished_blocks+=warp_len;
		}

		__syncwarp();

		last_diag = (i+_cudaSliceWidth)<<3;
		prev_max_score = query_len+ref_len-1;

		__syncwarp();

		// If job is finished
		if (terminated) { // This was for zdrop so maybe was never needed and never called?
			// this is to mark that the job is finished
			total_anti_diags = i; // set the total amount of diagonals as the current diagonal (to indicate that the job has finished)	
			if (warp_id==0) shared_job[(warp_num&3)] = total_anti_diags; //update this to shared memory as well (this will be used in Subwarp Rejoining as an indicator that the subwarp's job is done)
		}

		// Update the max score and its index to shared memory (used in Subwarp Rejoining)
		if (warp_id==1) shared_job[20+(warp_num&3)] = warp_num_block;

		__syncwarp();

		i += _cudaSliceWidth;

		/*Job wrap-up*/
		// If the job is done (either due to (1) meeting the termination condition (2) all the diagonals have been computed)
		if (i >= total_anti_diags) {

			// Spill the results to GPU memory to be later moved to the CPU
			if (warp_id==0) {
				device_res->aln_score[ub_idx] = global_max_score[warp_num_block];//copy the max score to the output array in the GPU mem
				device_res->query_batch_end[ub_idx] = global_max_query_idx[warp_num_block];//copy the end position on query_batch sequence to the output array in the GPU mem
				device_res->target_batch_end[ub_idx] = global_max_ref_idx[warp_num_block];//copy the end position on target_batch sequence to the output array in the GPU mem
				dropped[ub_idx] = terminated;
			}

			/*Subwarp Rejoining*/
			//The subwarp that has no job looks for new jobs by iterating over other subwarp's job
			for (m = 0; m < (32/const_warp_len); m++) {
				// if the selected job still has remainig diagonals
				if (shared_job[m] > i) { // possible because all subwarps sync after each diagonal block is finished
				// read the selected job's info
					total_anti_diags = shared_job[m];
					warp_num = ((warp_num>>2)<<2)+m;
					ub_idx = shared_job[16+m];

					packed_ref_batch_idx = shared_job[4+m];
					packed_query_batch_idx = shared_job[8+m];
					ref_len = shared_job[12+m];
					query_len = ref_len&65535;
					ref_len = ref_len>>16;
					packed_query_len = (query_len >> 3) + (query_len & 7 ? 1 : 0);
					packed_ref_len = (ref_len >> 3) + (ref_len & 7 ? 1 : 0);

					warp_num_block = shared_job[20+m];
					tb_offset = tb_ofs[ub_idx];
					
					// reset the flag
					terminated = dropped[ub_idx];
			
					break;
				}
			}

		}

		__syncwarp();

		/*Subwarp Rejoining*/
		//Set the mask, warp length and thread id within the warp 
		same_threads = __match_any_sync(__activemask(), warp_num);
		warp_len = __popc(same_threads);
		warp_id = __popc((((0xffffffff) << (threadIdx.x % 32))&same_threads))-1;

		__syncwarp();

		} // end of main alignment loop
		__syncwarp();
		/*Subwarp Rejoining*/
		//Reset subwarp and job related values for the next iteration
		warp_len = const_warp_len;
		warp_num = tid / warp_len;
		warp_id = tid % const_warp_len;
		ub_idx = shared_job[16+(warp_num&3)];
		warp_num_block = threadIdx.x / warp_len;

		__syncwarp();


	}

	return;


}


/* Gap right, left extension kernel (bw=1) */

#define KSW_CIGAR_MATCH  0
#define KSW_CIGAR_INS    1
#define KSW_CIGAR_DEL    2
#define KSW_CIGAR_N_SKIP 3

__device__
static inline void push_cigar(int *n_cigar, uint32_t *cigar, uint32_t op, int len)
{
	if (*n_cigar == 0 || op != (cigar[(*n_cigar) - 1]&0xf)) {
		cigar[(*n_cigar)++] = len<<4 | op;
	} else cigar[(*n_cigar)-1] += len<<4;
	return;
}

__global__
void mm2_kswz_extension(char* qseqs, char* tseqs, uint32_t* qseq_len, uint32_t* tseq_len, uint32_t* qseq_ofs, uint32_t* tseq_ofs,
						uint32_t* packed_tb_matrix, gasal_res_t *device_res, int* n_cigar, uint32_t* cigar, int n_task, uint32_t max_query_len, bool left){

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	uint8_t trace_dir[5000][3]; 

	for(int job_idx = tid; job_idx < n_task; job_idx += blockDim.x * gridDim.x){

		uint32_t qlen = qseq_len[tid];
		uint32_t tlen = tseq_len[tid];
		if(qlen <= 0 || tlen <= 0 || qlen > max_query_len || tlen > max_query_len) {
			n_cigar[tid] = 0; // TODO: also reset other part too & add dropped flag
			continue;
		}

		uint32_t qseq_offset = qseq_ofs[tid];
		uint32_t tseq_offset = tseq_ofs[tid];
		char* qseq = &qseqs[qseq_offset];
		char* tseq = &tseqs[tseq_offset];

		/* forward score computation */
		int r;

		// register for score computation
		int32_t h = MINUS_INF2; // current score - do we need this? maybe not...
		int32_t f; // f (from left)
		int32_t p[3] = {0,0,-(_cudaGapOE + _cudaGapExtend)}; // p (from diag)
		int32_t e[4] = {MINUS_INF2,- (_cudaGapOE * 2 + _cudaGapExtend),- (_cudaGapOE * 2 + _cudaGapExtend * 2),MINUS_INF2}; // e (from top)

		int32_t max_score = MINUS_INF2;
		uint32_t max_qpos = 0;
		uint32_t max_tpos = 0;

		// initialize registers

		for(int i = 0; i < tlen; i++){ // loop over tseq
			uint32_t tbase = tseq[i];
			f = (i==0)? - (_cudaGapOE * 2 + _cudaGapExtend) : MINUS_INF2; // initialize F : it comes from an out-of-bound cell
			e[3] = MINUS_INF2; // intialize rightmost E : it comes from an out-of-bound cell 
			for (int offset = -1; offset <= 1; offset++) {
				int j = i + offset;
				if (j < 0 || j >= qlen) continue;
				uint32_t qbase = qseq[j];

				int idx = offset + 1; 
				int e_idx = (idx + 1); // rolling window 
				int temp_score;
				
				// compute and update score
				DEV_GET_SUB_SCORE_GLOBAL(temp_score, qbase, tbase);
				temp_score += p[idx]; 
				uint32_t m_or_x = temp_score >= p[idx] ? 0 : 1;
				h = max(temp_score, f); 
				h = max(h, e[e_idx]); 
				trace_dir[i][idx] = 0;
				if(left){
					trace_dir[i][idx] |= (h == temp_score) ? m_or_x : ((f == temp_score) ? 3 : 2);
					trace_dir[i][idx] |= (temp_score - _cudaGapOE) > (f - _cudaGapExtend) ?  (uint32_t)0 : (uint32_t)1 << 3;
					trace_dir[i][idx] |= (temp_score - _cudaGapOE) > (e[e_idx] - _cudaGapExtend) ?  (uint32_t)0 : (uint32_t)1 << 2;
				}
				else {
					trace_dir[i][idx] |= h == temp_score ? m_or_x : (h == f ? (uint32_t)3 : (uint32_t)2); // 2 LSBs
					trace_dir[i][idx] |= (temp_score - _cudaGapOE) >= (f - _cudaGapExtend) ?  (uint32_t)0 : (uint32_t)1 << 3;
					trace_dir[i][idx] |= (temp_score - _cudaGapOE) >= (e[e_idx] - _cudaGapExtend) ?  (uint32_t)0 : (uint32_t)1 << 2;
				}
				f = max(temp_score- _cudaGapOE, f - _cudaGapExtend); // compute f for next (left) cell
				e[idx] = max(temp_score- _cudaGapOE, e[e_idx] - _cudaGapExtend); 
				p[idx] = h; 

				// update max score & check zdrop
				max_score = max(max_score, h);
				if(max_score == h){
					max_qpos = j;
					max_tpos = i;
				}
				// mm2_check_zdrop();
			}
		}
		device_res->aln_score[tid] = max_score;
	
		/* traceback phase */
		int i, j;
		i = max_tpos;
		j = max_qpos;

		if(i < 0 || j < 0 || i > max_query_len || j > max_query_len) return;
		int* n = &n_cigar[tid]; *n = 0;

		// traceback
		int state = 0;

		while (i >= 0 && j >= 0) {
			int direction = trace_dir[i][j-i+1];
			int force_state = -1;

			if(i > j + 1) force_state = 2; // bandwidth is 1
			if(i < j - 1) force_state = 3; 
	
			direction = force_state < 0? direction : 0;
	
			// for 1-stage gap cost
			if(state<=1) state = direction & 3;  
			else if(!(direction >> (state) & 1)) state = 0; 
			if(state<=1) state = direction & 3;  
			if (force_state >= 0) state = force_state; 
	
			switch(state) {
				case 0: // matched
				case 1: // mismatched
					push_cigar(n, &cigar[qseq_ofs[tid]], KSW_CIGAR_MATCH, 1);
					i--;
					j--;
				break;
				case 2: // from upper cell
					push_cigar(n, &cigar[qseq_ofs[tid]], KSW_CIGAR_DEL, 1);
					i--;
				break;
				case 3: // from left cell
					push_cigar(n, &cigar[qseq_ofs[tid]], KSW_CIGAR_INS, 1);
					j--;
				break;
				}
		}
		if (i >= 0) push_cigar(n, &cigar[qseq_ofs[tid]], KSW_CIGAR_DEL, i + 1); // first deletion
		if (j >= 0) push_cigar(n, &cigar[qseq_ofs[tid]], KSW_CIGAR_INS, j + 1); // first insertion
	
		if(!left){
			uint32_t tmp;
			for (i = 0; i < (*n)>>1; ++i) { // reverse CIGAR
				tmp = cigar[qseq_ofs[tid] + i];  // Store the original value
				cigar[qseq_ofs[tid] + i] = cigar[qseq_ofs[tid] + *n - 1 - i];  // Assign swapped value
				cigar[qseq_ofs[tid] + *n - 1 - i] = tmp;  // Complete the swap
			}
		}
	}
}


__global__
void mm2_kswz_extension_simd(char* qseqs, char* tseqs, uint32_t* qseq_len, uint32_t* tseq_len, uint32_t* qseq_ofs, uint32_t* tseq_ofs,
						uint32_t* packed_tb_matrix, gasal_res_t *device_res, int* n_cigar, uint32_t* cigar, int n_task, uint32_t max_query_len, bool left){
	// kernel to reproduce the minimap2 SIMD-operation
}